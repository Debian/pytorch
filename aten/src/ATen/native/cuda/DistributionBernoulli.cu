#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/ExpandUtils.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/CUDAGenerator.h>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/DistributionTemplates.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>
#include <utility>
#include <functional>

#include <ATen/native/Distributions.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/LegacyTHFunctionsCUDA.h>

#include <THC/THCGeneral.h>
#include <THC/THCApply.cuh>
#include <THC/THCDeviceUtils.cuh>

#include <cstdint>
#include <limits>
#include <utility>
#include <type_traits>

namespace {

template<typename scalar_t, typename prob_t>
void bernoulli_tensor_cuda_kernel(
    at::Tensor& ret, const at::Tensor& p,
    std::pair<uint64_t, uint64_t> seeds) {
  // The template argument `4` below indicates that we want to operate on four
  // element at each time. See NOTE [ CUDA_tensor_applyN helpers ] for details.
  at::cuda::CUDA_tensor_apply2<scalar_t, prob_t, 4>(
      ret, p,
      [seeds] __device__(
          int n, scalar_t& v1, scalar_t& v2, scalar_t& v3, scalar_t& v4,
          const prob_t& p1, const prob_t& p2, const prob_t& p3, const prob_t& p4) {
        hiprandStatePhilox4_32_10_t state;
        hiprand_init(
            seeds.first,
            blockIdx.x * blockDim.x + threadIdx.x,
            seeds.second,
            &state);
        // See Note [Register spilling in hiprand call for CUDA < 10]
        float4 rand = hiprand_uniform4(&state);
        switch (n) {
          case 4: {
            CUDA_KERNEL_ASSERT(0 <= p4 && p4 <= 1);
            v4 = static_cast<scalar_t>(rand.w <= p4);
            // fallthrough
          }
          case 3: {
            CUDA_KERNEL_ASSERT(0 <= p3 && p3 <= 1);
            v3 = static_cast<scalar_t>(rand.z <= p3);
            // fallthrough
          }
          case 2: {
            CUDA_KERNEL_ASSERT(0 <= p2 && p2 <= 1);
            v2 = static_cast<scalar_t>(rand.y <= p2);
            // fallthrough
          }
          case 1: {
            CUDA_KERNEL_ASSERT(0 <= p1 && p1 <= 1);
            v1 = static_cast<scalar_t>(rand.x <= p1);
          }
        }
      }
    );
}

} // namespace

namespace at { namespace native {

Tensor& bernoulli_tensor_cuda_(Tensor &self, const Tensor& p_, Generator* gen_) {
  NoNamesGuard guard;
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  std::pair<uint64_t, uint64_t> rng_engine_inputs;
  {
    // See Note [Acquire lock when using random generators]
    std::lock_guard<std::mutex> lock(gen->mutex_);
    rng_engine_inputs = gen->philox_engine_inputs(10);
  }
  auto p = std::get<0>(expand_inplace(self, p_.to(kCUDA)));
  AT_DISPATCH_ALL_TYPES_AND3(
    at::ScalarType::Half, at::ScalarType::BFloat16, at::ScalarType::Bool, self.scalar_type(), "bernoulli_tensor_cuda_self_", [&] {
      using self_t = scalar_t;
      AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, p.scalar_type(), "bernoulli_tensor_cuda_p_", [&] {
        using p_t = scalar_t;
        return bernoulli_tensor_cuda_kernel<self_t, p_t>(self, p, rng_engine_inputs);
      });
   });
  return self;
}

void bernoulli_scalar_cuda_kernel(TensorIterator& iter, double p_, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  AT_DISPATCH_ALL_TYPES_AND3(
    at::ScalarType::Half, at::ScalarType::BFloat16, at::ScalarType::Bool, iter.dtype(), "bernoulli_scalar_cuda_", [&] {
      if (std::is_same<scalar_t, double>::value) {
      // define lambda for bernoulli transformation
      auto bernoulli_func = [p_] __device__ (double rand) {
        return static_cast<scalar_t>(rand <= p_);
      };
      distribution_nullary_kernel<scalar_t, double, curand4_engine_calls/2>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_uniform2_double(state); },
        bernoulli_func);
    } else {
      auto p = static_cast<float>(p_);
      auto bernoulli_func = [p] __device__ (float rand) {
        return static_cast<scalar_t>(rand <= p);
      };
      distribution_nullary_kernel<scalar_t, float, curand4_engine_calls>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_uniform4(state); },
        bernoulli_func);
    }
   });
}

Tensor& bernoulli_scalar_cuda_(Tensor &self, double p, Generator* gen) {
  TORCH_CHECK(0 <= p && p <= 1, "bernoulli_ expects p to be in [0, 1], but got p=", p);
  auto iter = TensorIterator::nullary_op(self);
  bernoulli_scalar_cuda_kernel(iter, p, gen);
  return self;
}

}} // namespace at::native
