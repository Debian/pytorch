#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/zmath.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/BinaryOps.h>
#include <c10/macros/Macros.h>

// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at { namespace native {

void add_kernel_cuda(TensorIterator& iter, Scalar alpha_scalar) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(kHalf, kBool, kBFloat16, iter.common_dtype(), "add_cuda/sub_cuda", [&]() {
    using thrust_t = typename ztype_cuda<scalar_t>::thrust_t;
    auto alpha = thrust_t(alpha_scalar.to<scalar_t>());
    gpu_kernel_with_scalars(iter, [alpha]GPU_LAMBDA(thrust_t a, thrust_t b) -> thrust_t {
      return a + alpha * b;
    });
  });
}

static void sub_kernel_cuda(TensorIterator& iter, Scalar alpha_scalar) {
  add_kernel_cuda(iter, -alpha_scalar);
}

void div_kernel_cuda(TensorIterator& iter) {
  if (!isIntegralType(iter.common_dtype(), /*includeBool*/ false) && iter.is_cpu_scalar(2)) {
    // optimization for floating-point types: if the second operand is a CPU
    // scalar, compute a * reciprocal(b). Note that this may lose one bit of
    // precision compared to computing the division.
    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(kHalf, kBFloat16, iter.common_dtype(), "div_cuda", [&]() {
      using thrust_t = typename ztype_cuda<scalar_t>::thrust_t;
      auto inv_b = thrust_t(1.0) / thrust_t(iter.scalar_value<scalar_t>(2));
      iter.remove_operand(2);
      gpu_kernel(iter, [inv_b]GPU_LAMBDA(thrust_t a) -> thrust_t {
        return a * inv_b;
      });
    });
  } else {
    AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(kHalf, kBFloat16, iter.common_dtype(), "div_cuda", [&]() {
      using thrust_t = typename ztype_cuda<scalar_t>::thrust_t;
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(thrust_t a, thrust_t b) -> thrust_t {
        return a / b;
      });
    });
  }
}

void mul_kernel_cuda(TensorIterator& iter) {
  if (iter.common_dtype() == ScalarType::Bool) {
    // Workaround for the error: '*' in boolean context, suggest '&&' instead [-Werror=int-in-bool-context]
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(bool a, bool b) -> bool {
      return a && b;
    });
  } else {
    AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(kHalf, kBFloat16, iter.common_dtype(), "mul_cuda", [&]() {
      using thrust_t = typename ztype_cuda<scalar_t>::thrust_t;
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(thrust_t a, thrust_t b) -> thrust_t {
        return a * b;
      });
    });
  }
}

void remainder_kernel_cuda(TensorIterator& iter) {
  if (isIntegralType(iter.dtype(), /*includeBool*/ false)) {
    AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "remainder_cuda", [&]() {
      using thrust_t = typename ztype_cuda<scalar_t>::thrust_t;
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(thrust_t a, thrust_t b) -> thrust_t {
        thrust_t r = a % b;
        if ((r != 0) && ((r < 0) != (b < 0))) {
          r += b;
        }
        return r;
      });
    });
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "remainder_cuda", [&]() {
      using thrust_t = typename ztype_cuda<scalar_t>::thrust_t;
      gpu_kernel_with_scalars(iter,
        []GPU_LAMBDA(thrust_t a, thrust_t b) __ubsan_ignore_float_divide_by_zero__ -> thrust_t {
          return a - b * static_cast<thrust_t>(std::floor(a / b));
        });
    });
  }
}

REGISTER_DISPATCH(add_stub, &add_kernel_cuda);
REGISTER_DISPATCH(sub_stub, &sub_kernel_cuda);
REGISTER_DISPATCH(div_stub, &div_kernel_cuda);
REGISTER_DISPATCH(mul_stub, &mul_kernel_cuda);
REGISTER_DISPATCH(remainder_stub, &remainder_kernel_cuda);

}} // namespace at::native
