#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/Math.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/BinaryOps.h>


// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at { namespace native {

void atan2_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "atan2_cuda", [&]() {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return ::atan2(a, b);
    });
  });
}

void smooth_l1_kernel_cuda(TensorIterator& iter, double beta) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "smooth_l1_cuda", [&iter, beta]() {
    scalar_t beta_val(beta);
    gpu_kernel(iter, [beta_val] GPU_LAMBDA (scalar_t a, scalar_t b) -> scalar_t {
      auto z = ::abs(a - b);
      return z < beta_val ? scalar_t(0.5) * z * z / beta_val : z - scalar_t(0.5) * beta_val;
    });
  });
}


void mse_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "mse_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      auto diff = a - b;
      return diff * diff;
    });
  });
}

void logaddexp_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES(iter.dtype(), "logaddexp_cuda", [&]() {
    gpu_kernel(iter, [] GPU_LAMBDA (scalar_t a, scalar_t b) -> scalar_t {
      if (::isinf(a) && a == b) {
        return a;
      }
      else {
        scalar_t m = ::max(a, b);
        return m + ::log((scalar_t)(1.0) + ::exp(-::abs(a - b)));
      }
    });
  });
}

void logaddexp2_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES(iter.dtype(), "logaddexp2_cuda", [&]() {
    gpu_kernel(iter, [] GPU_LAMBDA (scalar_t a, scalar_t b) -> scalar_t {
      if (::isinf(a) && a == b) {
        return a;
      }
      else {
        scalar_t m = ::max(a, b);
        return m + ::log2((scalar_t)(1.0) + ::pow((scalar_t)(2.0), -::abs(a - b)));
      }
    });
  });
}

void gcd_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "gcd_cuda", [&]() {
    gpu_kernel(iter, [] GPU_LAMBDA (scalar_t a, scalar_t b) -> scalar_t {
      return calc_gcd(a, b);
    });
  });
}

void lcm_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "lcm_cuda", [&]() {
    gpu_kernel(iter, [] GPU_LAMBDA (scalar_t a, scalar_t b) -> scalar_t {
      scalar_t g = calc_gcd(a, b);
      return (g == 0) ? 0 : ::abs(a / g * b);
    });
  });
}

void hypot_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "hypot_cuda", [&]() {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return ::hypot(a, b);
    });
  });
}

void nextafter_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES(iter.common_dtype(), "nextafter_cuda", [&]() {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return ::nextafter(a, b);
    });
  });
}

void heaviside_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND3(kHalf, kBool, kBFloat16, iter.dtype(), "heaviside_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return a == 0 ? b : static_cast<scalar_t>(a > 0);
    });
  });
}

REGISTER_DISPATCH(atan2_stub, &atan2_kernel_cuda);
REGISTER_DISPATCH(smooth_l1_stub, &smooth_l1_kernel_cuda);
REGISTER_DISPATCH(mse_stub, &mse_kernel_cuda);
REGISTER_DISPATCH(logaddexp_stub, &logaddexp_kernel_cuda);
REGISTER_DISPATCH(logaddexp2_stub, &logaddexp2_kernel_cuda);
REGISTER_DISPATCH(gcd_stub, &gcd_kernel_cuda);
REGISTER_DISPATCH(lcm_stub, &lcm_kernel_cuda);
REGISTER_DISPATCH(hypot_stub, &hypot_kernel_cuda);
REGISTER_DISPATCH(nextafter_stub, &nextafter_kernel_cuda);
REGISTER_DISPATCH(heaviside_stub, &heaviside_kernel_cuda);

}} // namespace at::native
