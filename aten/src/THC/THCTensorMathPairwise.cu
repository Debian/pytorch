#include "hip/hip_runtime.h"
#include <THC/THCTensorMath.h>
#include <THC/THCGeneral.h>
#include <TH/THHalf.h>
#include <THC/THCTensorCopy.h>
#include <THC/THCApply.cuh>
#include <THC/THCNumerics.cuh>
#include <THC/THCTensorMathCompareT.cuh>
#include <THC/THCTensor.hpp>

template <typename T>
struct TensorMulConstantOp {
  TensorMulConstantOp(T v) : val(v) {}
  __device__ __forceinline__ void operator()(T* out, T* in) {
    *out = *in * val;
  }

  __device__ __forceinline__ void operator()(T* v) {
    *v *= val;
  }

  const T val;
};

template <typename T>
struct TensorFmodOp {
  TensorFmodOp(T v) : val((float)v) {}
  __device__ __forceinline__ void operator()(T* out, T* in) {
    *out = (T) fmodf((float) *in, val);
  }

  __device__ __forceinline__ void operator()(T* v) {
    *v = (T) fmodf((float) *v, val);
  }

  const float val;
};

template <>
struct TensorFmodOp<double> {
  TensorFmodOp(double v) : val(v) {}
  __device__ __forceinline__ void operator()(double* out, double* in) {
    *out = fmod(*in, val);
  }

  __device__ __forceinline__ void operator()(double* v) {
    *v = fmod(*v, val);
  }

  const double val;
};

#include <THC/generic/THCTensorMathPairwise.cu>
#include <THC/THCGenerateAllTypes.h>

#include <THC/generic/THCTensorMathPairwise.cu>
#include <THC/THCGenerateBoolType.h>
