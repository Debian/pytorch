#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/ExpandUtils.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/CUDAGenerator.h>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/DistributionTemplates.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>
#include <utility>
#include <functional>

#include <ATen/native/Distributions.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/LegacyTHFunctionsCUDA.h>

#include <THC/THCGeneral.h>
#include <THC/THCApply.cuh>
#include <THC/THCDeviceUtils.cuh>

#include <cstdint>
#include <limits>
#include <utility>
#include <type_traits>

namespace at { namespace native {

void cauchy_kernel(TensorIterator& iter, double median_, double sigma_, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "cauchy_cuda", [&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    auto median = static_cast<accscalar_t>(median_);
    auto sigma = static_cast<accscalar_t>(sigma_);
    if (std::is_same<scalar_t, double>::value) {
      // define lambda for cauchy transformation
      auto cauchy_func = [median, sigma] __device__ (accscalar_t rand) {
        return static_cast<scalar_t>(median + sigma *
                ::tan(static_cast<accscalar_t>(M_PI) * (rand-static_cast<accscalar_t>(0.5))));
      };
      distribution_nullary_kernel<scalar_t, accscalar_t, curand4_engine_calls/2>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_uniform2_double(state); },
        cauchy_func);
    } else {
      // use __tanf fast approximation for peak bandwidth
      auto cauchy_func = [median, sigma] __device__ (accscalar_t rand) {
        return static_cast<scalar_t>(median + sigma *
                __tanf(static_cast<accscalar_t>(M_PI) * (rand-static_cast<accscalar_t>(0.5))));
      };
      distribution_nullary_kernel<scalar_t, accscalar_t, curand4_engine_calls>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_uniform4(state); },
        cauchy_func);
    }
   });
}

REGISTER_DISPATCH(cauchy_stub, &cauchy_kernel);

}} // namespace at::native
