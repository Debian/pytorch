#include "hip/hip_runtime.h"
#include <ATen/native/TensorAdvancedIndexing.h>
#include <ATen/native/IndexingUtils.h>

#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/ExpandUtils.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/detail/IndexUtils.cuh>

#include <THC/THCDeviceUtils.cuh>
#include <THC/THCGeneral.h>
#include <THC/THCTensorSort.cuh>
#include <ATen/cuda/HIPContext.h>
#include <THC/THCThrustAllocator.cuh>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>
#include <THC/THCAtomics.cuh>

#include <c10/macros/Macros.h>

namespace {

template <typename scalar_t, int SZ>
__global__ void indexing_backward_kernel(
  int64_t* sorted_indices, int64_t* indices, scalar_t* grad_output, scalar_t* grad_weight,
  int64_t numel, int64_t stride, int64_t stride_before, int64_t outer_dim) {
//numel is total number of flattened indices, not expanded to dimensions that are not indexed.
//stride is the cumulative size of the not-indexed last dimensions
//stride_before is the stride of the dimension immediately preceding first indexed dimension
//if indexing starts from the 0th dimension, stride_before does not matter because blockIdx.z will be 0 in this case
//outer_dim is number of elements in the first unindexed dimensions
  using accscalar_t = at::acc_type<scalar_t, true>;

  // Each warp is responsible for an input into the LookupTable.
  // If the preceding input has the same destination index as this input, then the warp
  // exits immediately. The warp also processes subsequent inputs with the
  // same value.
  //
  // Input Warp
  // 1     <warp 1>
  // 1     <warp 1> (<warp 2> exits without doing any work)
  // 5     <warp 3>
  // 8     <warp 4>

  // Number of values processed by each thread (grain size)
  for (int64_t z = blockIdx.z; z < outer_dim; z += gridDim.z){
    int64_t idx = blockIdx.x * blockDim.y + threadIdx.y;
    if (idx < numel
        && (idx == 0 || sorted_indices[idx] != sorted_indices[idx - 1])){
      do {
        int64_t start_feature = threadIdx.x + blockIdx.y * blockDim.x * SZ;
        const int64_t weight_row = ((int64_t) sorted_indices[idx]) * stride + z * stride_before;
        const int64_t grad_row = ((int64_t) indices[idx]) * stride + z * numel * stride;
        const accscalar_t scale = (accscalar_t)1.0;

        accscalar_t gradient[SZ];
        accscalar_t weight[SZ];

        while (start_feature < stride) {
          #pragma unroll
          for (int ii = 0; ii < SZ; ii++) {
            int64_t feature_dim = start_feature + ii * C10_WARP_SIZE;
            if (feature_dim < stride) {
              gradient[ii] = static_cast<accscalar_t>(grad_output[grad_row + feature_dim]);
              weight[ii] = static_cast<accscalar_t>(grad_weight[weight_row + feature_dim]);
            }
          }

          #pragma unroll
          for (int ii = 0; ii < SZ; ii++) {
            weight[ii] += gradient[ii] * scale;
          }

          #pragma unroll
          for (int ii = 0; ii < SZ; ii++) {
            int64_t feature_dim = start_feature + ii * C10_WARP_SIZE;
            if (feature_dim < stride) {
                grad_weight[weight_row + feature_dim] = static_cast<scalar_t>(weight[ii]);
            }
          }
          start_feature += gridDim.y * blockDim.x * SZ;
        }

        idx++;
      } while (idx < numel && sorted_indices[idx] == sorted_indices[idx - 1]);
    }
  }
}


}


namespace at { namespace native {

static Tensor wrapIndexOnce(const Tensor & index, int64_t dim, int64_t dim_size, bool check_range=true) {
//we don't need to check range in backward - if there were out of bounds indices forward should already have errored out
  if (index.numel() != 0 && check_range) {
    auto max_idx = index.max().item<int64_t>();
    auto min_idx = index.min().item<int64_t>();
    if (max_idx >= dim_size) {
      TORCH_CHECK_INDEX(false, "index ", max_idx, " is out of bounds for dimension ", dim, " with size ", dim_size);
    }
    if (min_idx < -dim_size) {
      TORCH_CHECK_INDEX(false, "index ", min_idx, " is out of bounds for dimension ", dim, " with size ", dim_size);
    }
  }
  return index.remainder(dim_size);
}

static std::vector<int64_t> computeLinearStride(const Tensor & tensor) {
  // computes the stride as if tensor were contiguous
  auto sizes = tensor.sizes();
  std::vector<int64_t> stride(tensor.dim());
  stride[tensor.dim() - 1] = 1;
  std::partial_sum(sizes.rbegin(), sizes.rend() - 1, stride.rbegin() + 1, std::multiplies<int64_t>());
  return stride;
}

static std::tuple<Tensor, int64_t, int64_t, int64_t>
computeLinearIndex(const Tensor & src, TensorList indices, bool check_range) {
  auto strides = computeLinearStride(src);
  const auto& backend = src.type().backend();

  // Compute the linear index by multiplying the indexing tensors by the
  // stride and summing them. All the indexing tensors have the same shape at
  // this point. We also compute the number of dimensions before and after that
  // are not being index.
  Tensor linearIndex;
  int64_t emptyBefore = 0, emptyAfter = 0, nElemBefore = 1, nElemAfter = 1, strideBefore =0;
  for (auto i = decltype(src.dim()){0}; i < src.dim(); i++) {
    if (indices[i].defined()) {
      // Cast index to the longType matching src's backend
      // This allows us to support ie indexing a cuda tensor with a cpu tensor
      Tensor index = (wrapIndexOnce(indices[i], i, src.size(i), check_range) * strides[i]).toBackend(backend);
      if (linearIndex.defined()) {
        linearIndex += index;
      } else {
        linearIndex = index;
        if (i>0) {
           strideBefore = src.stride(i-1); // stride after undefined dimensions
        }
      }
    } else if (linearIndex.defined()) {
      emptyAfter++;
      nElemAfter *= src.size(i);
    } else {
      emptyBefore++;
      nElemBefore *= src.size(i);
    }
  }

  return std::make_tuple(std::move(linearIndex), nElemBefore, strideBefore, nElemAfter);
}


static std::tuple<Tensor, Tensor, int64_t, int64_t, int64_t, std::vector<int64_t>> makeLinearIndex(Tensor self, TensorList orig, bool check_range) {
  checkIndexTensorTypes(orig);
  // first expand BoolTensor (masks) or ByteTensor (masks) into 1 or more LongTensors
  auto indices = expandTensors(self, orig);
  // next broadcast all index tensors together
  indices = expand_outplace(indices);
  // add missing null Tensors so that it matches self.dim()
  while (indices.size() < (size_t)self.dim()) {
    indices.emplace_back();
  }
  // if the non-null indices are not all adjacent, transpose self and indices
  // together so that they're adjacent at the front
  std::vector<int64_t> inversePerm;
  if (!hasContiguousSubspace(indices)) {
    std::tie(self, indices, inversePerm) = transposeToFrontAndInvPerm(self, indices);
  }
  int64_t nElemBefore, strideBefore, nElemAfter;
  Tensor linearIndex;
  std::tie(linearIndex, nElemBefore, strideBefore, nElemAfter) = computeLinearIndex(self, indices, check_range);
  return std::make_tuple(linearIndex, self, nElemBefore, strideBefore, nElemAfter, inversePerm);
}


namespace {
void index_put_accum_kernel(Tensor & self, TensorList indices, const Tensor & value, bool unsafe) {
  if (indices.size() > (size_t)self.dim()) {
    TORCH_CHECK_INDEX(false, "too many indices for tensor of dimension ", self.dim(), " (got ", indices.size(), ")");
  }
  auto value_ = value.contiguous();
  Tensor linearIndex, expandedValue, src;
  int64_t nElemBefore, strideBefore, sliceSize;
  std::vector<int64_t> inversePerm;
  std::tie(linearIndex, src, nElemBefore, strideBefore, sliceSize, inversePerm) = makeLinearIndex(self, indices, !unsafe);
  int64_t num_indices = linearIndex.numel();
  if (num_indices > 0 && sliceSize > 0) {
      const bool permuted = !src.is_contiguous();
      auto src_ = permuted ? src.contiguous() : src;
      linearIndex = linearIndex.reshape(-1);
      auto sorted_indices = at::empty_like(linearIndex, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
      auto orig_indices = at::empty_like(linearIndex, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
      using device_ptr = thrust::device_ptr<int64_t>;
      const hipStream_t stream = at::cuda::getCurrentCUDAStream();

      linearIndex.floor_divide_(sliceSize);
      {
      sorted_indices.copy_(linearIndex);
      auto allocator = THCThrustAllocator(globalContext().lazyInitCUDA());
      auto policy = thrust::cuda::par(allocator).on(stream);

      // Fill sortedOrigIndices with sequential indices
      const auto count_iter = thrust::counting_iterator<int64_t>(0);
      auto orig_data = device_ptr(orig_indices.data_ptr<int64_t>());
      thrust::copy(policy, count_iter, count_iter + num_indices, orig_data);

      // Sort the inputs into sorted with the corresponding indices; we
      // don't need a stable or multidimensional sort, so just use Thrust
      // directly
      // Sort; a stable sort is not required
      // NB - not passing comparator causes thrust to use radix sort, and it hurts perf A LOT, at least for medium (few K) sized indices
      auto sorted_data = device_ptr(sorted_indices.data_ptr<int64_t>());
      thrust::sort_by_key(policy, sorted_data, sorted_data + num_indices, orig_data, ThrustLTOp<int64_t>());
      }
      TORCH_INTERNAL_ASSERT(linearIndex.numel()*sliceSize*nElemBefore == value.numel(), "number of flattened indices did not match number of elements in the value tensor", linearIndex.numel()*sliceSize*nElemBefore, value.numel());
      const int UNROLL = 4;
      const int indices_per_block = 4;
      dim3 grid(THCCeilDiv(num_indices, (int64_t) indices_per_block),
           std::min<int>(at::cuda::getCurrentDeviceProperties()->maxGridSize[1], THCCeilDiv(sliceSize, (int64_t) (C10_WARP_SIZE*UNROLL))),
           std::min(std::max<int>(1,nElemBefore), at::cuda::getCurrentDeviceProperties()->maxGridSize[2]));
      dim3 block(C10_WARP_SIZE, indices_per_block);

      AT_DISPATCH_ALL_TYPES_AND3(at::ScalarType::Half, at::ScalarType::Bool, at::ScalarType::BFloat16,
      value_.scalar_type(), "indexing_backward", [&] {
      AT_SKIP_BFLOAT16_IF_NOT_ROCM(scalar_t, "indexing_backward", [&] {
      indexing_backward_kernel<scalar_t, UNROLL><<<grid, block, 0, stream>>>(
        sorted_indices.data_ptr<int64_t>(),
        orig_indices.data_ptr<int64_t>(),
        value_.data_ptr<scalar_t>(),
        src_.data_ptr<scalar_t>(),
        num_indices,
        sliceSize,
        strideBefore,
        nElemBefore);
      });
      });
      AT_CUDA_CHECK(hipGetLastError());
      if (permuted)
          self.copy_(src_.permute(inversePerm));
  }
}

REGISTER_CUDA_DISPATCH(index_put_accum_stub, &index_put_accum_kernel);
} //anonymous


// Check tensor dimensions for index operations, and return the slice size.
static ptrdiff_t getSliceSize(const Tensor & dst,
                              int dim,
                              const Tensor & index,
                              const Tensor & src)
{
  int dstDims = dst.dim();
  int srcDims = src.dim();

  TORCH_CHECK(index.dim() <= 1, "Index must be vector or scalar");

  ptrdiff_t dstSliceSize = 1;
  TORCH_CHECK(dim >= 0 && dim < dstDims, "Indexing dim ", dim, " is out of bounds");
  for (int d = 0; d < dstDims; d++) {
    if (d != dim) {
      dstSliceSize *= dst.size(d);
    }
  }

  TORCH_CHECK(dim < srcDims, "Indexing dim ", dim, " is out of bounds");
  TORCH_CHECK(index.numel() == src.size(dim),
             "length of src.size[dim] is not equal to length of indices");

  ptrdiff_t srcSliceSize = 1;
  bool mismatch = false;

  if (dstDims != srcDims) mismatch = true;

  for (int d = 0; d < srcDims; d++) {
    if (d != dim) {
      srcSliceSize *= src.size(d);
      if (!mismatch && dst.size(d) != src.size(d)) mismatch = true;
    }
  }

  TORCH_CHECK(dstSliceSize == srcSliceSize,
             "Source/destination tensor have different slice sizes (%ld vs %ld)",
             dstSliceSize, srcSliceSize);

  if (mismatch) {
    TORCH_WARN_ONCE(
        "Warning: source/destination slices have same size but different "
        "shape for an index operation.  This behavior is deprecated.\n");
  }

  return dstSliceSize;
}

// We prefer this kernel to avoid reloading index points if the number
// of indices is a small number.
// This kernel in fact works for all choices of problem size, but if
// the number of indices chosen is large, then the
// indexAddLargeIndex kernel is a better choice to increase
// parallelism.
template <typename T, typename IndexType, int DstDim, int SrcDim, int IdxDim>
__global__ void indexAddSmallIndex(cuda::detail::TensorInfo<T, IndexType> dst,
                                   cuda::detail::TensorInfo<T, IndexType> src,
                                   cuda::detail::TensorInfo<int64_t, IndexType> indices,
                                   int dstAddDim,
                                   int srcAddDim,
                                   IndexType innerSize,
                                   int64_t dstAddDimSize) {
  // In order to avoid reloading the index that we are copying, load
  // it once to handle all of the points that are being selected, so
  // it can be reused as much as possible. This kernel is chosen when
  // this is a good choice (small number of chosen indices), since
  // re-accessing indices in addition to src elements can be slow.
  for (IndexType srcIndex = 0; srcIndex < indices.sizes[0]; ++srcIndex) {
    // Lua indices begin at 1
    IndexType dstIndex =
        indices.data[cuda::detail::IndexToOffset<int64_t, IndexType, IdxDim>::get(srcIndex, indices)];
    CUDA_KERNEL_ASSERT(dstIndex < dstAddDimSize);

    // We stride over the output ignoring the indexed dimension
    // (innerSize), whose offset calculation is handled differently
    for (IndexType linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
         linearIndex < innerSize;
         linearIndex += gridDim.x * blockDim.x) {
      IndexType dstOffset =
          cuda::detail::IndexToOffset<T, IndexType, DstDim>::get(linearIndex, dst);
      dstOffset += dstIndex * dst.strides[dstAddDim];

      IndexType srcOffset =
          cuda::detail::IndexToOffset<T, IndexType, SrcDim>::get(linearIndex, src);
      srcOffset += srcIndex * src.strides[srcAddDim];

      gpuAtomicAdd(&dst.data[dstOffset], src.data[srcOffset]);
    }
  }
}

// We prefer this kernel to balance parallelism across index points,
// if there are a large number of indices.
// This kernel in fact works for all choices of problem size, but if
// the number of indices chosen is small, then the
// indexAddSmallIndex kernel is a better choice to reduce memory
// accesses.
template <typename T, typename IndexType, int DstDim, int SrcDim, int IdxDim,
          bool IndexIsMajor>
__global__ void indexAddLargeIndex(cuda::detail::TensorInfo<T, IndexType> dst,
                                   cuda::detail::TensorInfo<T, IndexType> src,
                                   cuda::detail::TensorInfo<int64_t, IndexType> indices,
                                   int dstAddDim,
                                   int srcAddDim,
                                   IndexType totalSize,
                                   IndexType innerSize,
                                   int64_t dstAddDimSize) {
  // We stride over the output including the indexed dimension
  // (totalSize), and calculate the destination index point based on that
  for (IndexType linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
       linearIndex < totalSize;
       linearIndex += gridDim.x * blockDim.x) {
    IndexType srcIndex, elementInSlice;
    if (IndexIsMajor) {
      srcIndex = linearIndex / innerSize;
      elementInSlice = linearIndex % innerSize;
    }
    else {
      elementInSlice = linearIndex / innerSize;
      srcIndex = linearIndex % innerSize;
    }

    // Lua indices begin at 1
    IndexType dstIndex =
        indices.data[cuda::detail::IndexToOffset<int64_t, IndexType, IdxDim>::get(srcIndex, indices)];
    CUDA_KERNEL_ASSERT(dstIndex < dstAddDimSize);

    IndexType dstOffset =
      cuda::detail::IndexToOffset<T, IndexType, DstDim>::get(elementInSlice, dst);
    dstOffset += dstIndex * dst.strides[dstAddDim];

    IndexType srcOffset =
      cuda::detail::IndexToOffset<T, IndexType, SrcDim>::get(elementInSlice, src);
    srcOffset += srcIndex * src.strides[srcAddDim];

    gpuAtomicAdd(&dst.data[dstOffset], src.data[srcOffset]);
  }
}

// Compare the stride between adjacent slices (sliceStride) with strides in the
// other dimensions (i.e., strides *inside* each slice).
//
// - Returns true if some dimension inside the slice has lower stride than
//   sliceStride.  The simplest example is a 2-D contiguous tensor with sliceDim
//   == 0 (that is, each slice is a row).
//
//   In this case, we choose the CUDA kernel that processes the data in
//   "index-major order".  For example, if thread count equals slice size, then
//   all threads process slice #0 in lockstep, and then slice #1, and so on.
//
// - Otherwise (i.e., sliceStride has the lowest value), this function returns
//   false.  The simplest example is a 2-D contiguous tensor with sliceDim == 1
//   (each slice is a column).
//
//   In this case, we choose the CUDA kernel that processes the data in
//   "elementInSlice-major order".  For example, each thread can process element
//   #0 of every slice, and then element #1 of every slice, and so on.
template <typename scalar_t>
bool indexShouldBeMajor(cuda::detail::TensorInfo<scalar_t, unsigned int> &info,
                                    int sliceDim)
{
  // The stride between adjacent slices (e.g., between element #0 of slice #100
  // and element #0 of slice #101).
  unsigned int sliceStride = info.strides[sliceDim];

  for (int i = 0; i < info.dims; ++i) {
    if (i != sliceDim && info.sizes[i] > 1 && info.strides[i] < sliceStride) {
      return true;
    }
  }

  return false;
}

Tensor& index_add_cuda_(Tensor & self, int64_t dim, const Tensor & index, const Tensor & source) {
  dim = maybe_wrap_dim(dim, self.dim());

  TensorArg self_arg{self, "self", 1}, index_arg{index, "index", 3}, source_arg{source, "source", 4};
  checkAllSameGPU("index_add", {self_arg, index_arg, source_arg});

  TORCH_CHECK_INDEX(index.dim() <= 1, "index_add_(): Index is supposed to be a vector");
  TORCH_CHECK(index.scalar_type() == ScalarType::Long, "index_add_(): Expected dtype int64 for index");
  TORCH_CHECK(self.scalar_type() == source.scalar_type(),
              "index_add_(): self and source must have the same scalar type");
  TORCH_CHECK(dim == 0 || dim < source.dim(),
              "index_add_(): Indexing dim ", dim, " is out of bounds of tensor");
  TORCH_CHECK(index.numel() == (source.dim() == 0 ? 1 : source.size(dim)),
              "index_add_(): Number of indices should be equal to self.size(dim)");

  // Scalars are treated as 1-d tensor
  Tensor self_ = (self.dim() == 0) ? self.view(1) : self;
  Tensor source_ = (source.dim() == 0) ? source.view(1) : source;

  TORCH_CHECK(self.dim() <= MAX_CUTORCH_DIMS, CUTORCH_DIM_WARNING);
  TORCH_CHECK(source.dim() <= MAX_CUTORCH_DIMS, CUTORCH_DIM_WARNING);
  TORCH_CHECK(index.dim() <= MAX_CUTORCH_DIMS, CUTORCH_DIM_WARNING);

  // The `source` is partitioned into two parts:
  // -the size of each slice we are indexing, which is the
  // total size of the tensor ignoring dimension `dim`;
  // -the number of index we are choosing, which is the total size
  // of the tensor `index`.
  ptrdiff_t sliceSize = getSliceSize(self_, dim, index, source_);
  ptrdiff_t sourceTotalSize = source.numel();
  int64_t selfAddDimSize = self_.size(dim);
  ptrdiff_t numIndex = index.numel();

  if (sliceSize == 0) {
    return self;
  }
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  bool indContig = index.is_contiguous();

  int mpc = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;

#define SMALL_INDEX(TENSOR_TYPE, TYPE, SELF_DIM, SOURCE_DIM, IDX_DIM) \
  indexAddSmallIndex<TENSOR_TYPE, TYPE, SELF_DIM, SOURCE_DIM, IDX_DIM> \
    <<<smallIndexGrid, smallIndexBlock, 0, stream>>>(   \
      selfInfo, sourceInfo, indexInfo,                    \
      selfAddDim, sourceAddDim, sliceSize, selfAddDimSize);

#define LARGE_INDEX(TENSOR_TYPE, TYPE,                        \
                    SELF_DIM, SOURCE_DIM, IDX_DIM, IDX_IS_MAJOR)  \
  indexAddLargeIndex<TENSOR_TYPE, TYPE,                       \
                     SELF_DIM, SOURCE_DIM, IDX_DIM, IDX_IS_MAJOR> \
    <<<largeIndexGrid, largeIndexBlock, 0, stream>>>(         \
      selfInfo, sourceInfo, indexInfo,                          \
      selfAddDim, sourceAddDim, sourceTotalSize,                     \
      (IDX_IS_MAJOR) ? sliceSize : numIndex,                \
      selfAddDimSize);

  dim3 smallIndexGrid(std::min(THCCeilDiv(sliceSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 smallIndexBlock(std::min(sliceSize, (ptrdiff_t)128));

  dim3 largeIndexGrid(std::min(THCCeilDiv(sourceTotalSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 largeIndexBlock(std::min(sourceTotalSize, (ptrdiff_t)128));

  if (cuda::detail::canUse32BitIndexMath(self) &&
      cuda::detail::canUse32BitIndexMath(source) &&
      cuda::detail::canUse32BitIndexMath(index)) {
    AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, self.scalar_type(), "index_add", [&] {
      AT_SKIP_BFLOAT16_IF_NOT_ROCM(scalar_t, "index_add", [&] {
        cuda::detail::TensorInfo<scalar_t, unsigned int> selfInfo =
            cuda::detail::getTensorInfo<scalar_t, unsigned int>(self_);
        int selfAddDim = selfInfo.collapseDims(dim);
        selfInfo.reduceDim(selfAddDim);

        auto sourceInfo =
          cuda::detail::getTensorInfo<scalar_t, unsigned int>(source_);
        int sourceAddDim = sourceInfo.collapseDims(dim);
        sourceInfo.reduceDim(sourceAddDim);

        auto indexInfo =
         cuda::detail::getTensorInfo<int64_t, unsigned int>(index);
        indexInfo.collapseDims();

        // A reasonable choice for when to have each thread iterate over
        // index to choose
        if (numIndex <= 16) {
          if (selfInfo.dims == 1 && sourceInfo.dims == 1 && indContig) {
            SMALL_INDEX(scalar_t, unsigned int, 1, 1, -2);
          } else if (selfInfo.dims == 2 && sourceInfo.dims == 2 && indContig) {
            SMALL_INDEX(scalar_t, unsigned int, 2, 2, -2);
          } else if (selfInfo.dims == 3 && sourceInfo.dims == 3 && indContig) {
            SMALL_INDEX(scalar_t, unsigned int, 3, 3, -2);
          } else {
            SMALL_INDEX(scalar_t, unsigned int, -1, -1, -1);
          }
        } else {
          bool indexIsMajor = indexShouldBeMajor(selfInfo, selfAddDim);

          if (selfInfo.dims == 1 && sourceInfo.dims == 1 && indContig) {
            LARGE_INDEX(scalar_t, unsigned int, 1, 1, -2, true);
          } else if (selfInfo.dims == 2 && sourceInfo.dims == 2 && indContig) {
            if (indexIsMajor) {
              LARGE_INDEX(scalar_t, unsigned int, 2, 2, -2, true);
            } else {
              LARGE_INDEX(scalar_t, unsigned int, 2, 2, -2, false);
            }
          } else if (selfInfo.dims == 3 && sourceInfo.dims == 3 && indContig) {
            if (indexIsMajor) {
              LARGE_INDEX(scalar_t, unsigned int, 3, 3, -2, true);
            } else {
              LARGE_INDEX(scalar_t, unsigned int, 3, 3, -2, false);
            }
          } else {
            LARGE_INDEX(scalar_t, unsigned int, -1, -1, -1, true);
          }
        }
      });
    });
  } else {
    AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, self.scalar_type(), "index_add", [&] {
      AT_SKIP_BFLOAT16_IF_NOT_ROCM(scalar_t, "index_add", [&] {
        cuda::detail::TensorInfo<scalar_t, uint64_t> selfInfo =
          cuda::detail::getTensorInfo<scalar_t, uint64_t>(self_);
        int selfAddDim = selfInfo.collapseDims(dim);
        selfInfo.reduceDim(selfAddDim);

        cuda::detail::TensorInfo<scalar_t, uint64_t> sourceInfo =
          cuda::detail::getTensorInfo<scalar_t, uint64_t>(source_);
        int sourceAddDim = sourceInfo.collapseDims(dim);
        sourceInfo.reduceDim(sourceAddDim);

        cuda::detail::TensorInfo<int64_t, uint64_t> indexInfo =
          cuda::detail::getTensorInfo<int64_t, uint64_t>(index);
        indexInfo.collapseDims();

        LARGE_INDEX(scalar_t, uint64_t, -1, -1, -1, true);
      });
    });
  }

  return self;
#undef SMALL_INDEX
#undef LARGE_INDEX
}

} //at
} //native
