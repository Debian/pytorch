#include "hip/hip_runtime.h"
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Reduce.cuh>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/SharedReduceOps.h>
#include <ATen/Dispatch.h>
#include <ATen/native/ReduceOps.h>

namespace at { namespace native {

template <typename scalar_t, typename acc_t=scalar_t, typename out_t=scalar_t>
void norm_kernel_cuda_impl(TensorIterator& iter, Scalar val) {
  float p;
  if (val.isIntegral(false)) {
     p = val.to<int64_t>();
  } else if (val.isFloatingPoint()) {
     p = val.to<acc_t>();
  } else {
     AT_ERROR("norm_kernel_cuda_impl expects norm to be integer or float");
  }

  if (p == static_cast<float>(0)) {
    gpu_reduce_kernel<scalar_t, out_t>(iter, NormZeroOps<acc_t>(), 0);
  } else if (p == static_cast<float>(1)) {
    gpu_reduce_kernel<scalar_t, out_t>(iter, NormOneOps<acc_t>(), 0);
  } else if (p == static_cast<float>(2)) {
    gpu_reduce_kernel<scalar_t, out_t>(iter, NormTwoOps<acc_t>(), 0);
  } else if (p == static_cast<float>(INFINITY)) {
    gpu_reduce_kernel<scalar_t, out_t>(iter, AbsMaxOps<acc_t>(), std::numeric_limits<acc_t>::min());
  } else if (p == static_cast<float>(-INFINITY)) {
    gpu_reduce_kernel<scalar_t, out_t>(iter, AbsMinOps<acc_t>(), std::numeric_limits<acc_t>::max());
  } else {
    gpu_reduce_kernel<scalar_t, out_t>(iter, NormOps<acc_t>{ acc_t(p) }, 0);
  }
}

static void norm_kernel_cuda(TensorIterator& iter, Scalar p) {
  if (iter.dtype() == kHalf) {
    return norm_kernel_cuda_impl<at::Half, float>(iter, p);
  } else if (iter.dtype(1) == kHalf && iter.dtype() == kFloat) {
    // type promotion that does cast and reduction in a single kernel
    return norm_kernel_cuda_impl<at::Half, float, float>(iter, p);
  }
  #ifdef __HIP_PLATFORM_HCC__
  else if(iter.dtype() == kBFloat16) {
    return norm_kernel_cuda_impl<at::BFloat16, float>(iter, p);
  } else if (iter.dtype(1) == kBFloat16 && iter.dtype() == kFloat) {
    // type promotion that does cast and reduction in a single kernel
    return norm_kernel_cuda_impl<at::BFloat16, float, float>(iter, p);
  }
  #endif
  AT_DISPATCH_FLOATING_TYPES(iter.dtype(), "norm_cuda", [&]() {
    norm_kernel_cuda_impl<scalar_t>(iter, p);
  });
}

REGISTER_DISPATCH(norm_stub, &norm_kernel_cuda);

}} // namespace at::native
