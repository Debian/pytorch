#include "hip/hip_runtime.h"
#include <ATen/Context.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/Dispatch.h>
#include <ATen/NativeFunctions.h>

#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>

namespace at {
namespace native {

// ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~ triu/tril ~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~

template <typename scalar_t, typename IndexType, bool upper>
#ifdef __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_1(512)
#endif
__global__
void triu_tril_kernel(
    cuda::detail::TensorInfo<scalar_t, IndexType> result_info,
    const cuda::detail::TensorInfo<scalar_t, IndexType> self_info,
    const int64_t k, const int64_t N) {
  int64_t linear_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (linear_idx >= N) {
    return;
  }

  auto dims = self_info.dims;

  IndexType self_offset = 0, result_offset = 0;
  // Compute column index and corresponding offset
  IndexType col = linear_idx % self_info.sizes[dims - 1];
  linear_idx /= self_info.sizes[dims - 1];
  self_offset += self_info.strides[dims - 1] * col; 
  result_offset += result_info.strides[dims - 1] * col;

  // Compute row index and corresponding offset
  IndexType row = linear_idx % self_info.sizes[dims - 2];
  linear_idx /= self_info.sizes[dims - 2];
  self_offset += self_info.strides[dims - 2] * row;
  result_offset += result_info.strides[dims - 2] * row;

  // Compute remaining offsets
  IndexType running_index;
  #pragma unroll
  for (IndexType i = dims - 3; i >= 0; --i) {
    running_index = linear_idx % self_info.sizes[i];
    linear_idx /= self_info.sizes[i];
    self_offset += running_index * self_info.strides[i];
    result_offset += running_index * result_info.strides[i];
  }

  bool mask = upper ? (col - row >= k) : (col - row <= k);
  result_info.data[result_offset] = mask ? self_info.data[self_offset] : scalar_t(0);
}

template <bool upper>
Tensor& triu_tril_cuda_template(Tensor& result, const Tensor& self, int64_t k, const char* name) {
  int64_t N = self.numel();
  dim3 dim_block = cuda::getApplyBlock();
  dim3 dim_grid((N + dim_block.x - 1) / dim_block.x);
  AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::Bool, self.scalar_type(), name, [&]{
    if (cuda::detail::canUse32BitIndexMath(result) && cuda::detail::canUse32BitIndexMath(self)) {
      auto result_info = cuda::detail::getTensorInfo<scalar_t, int32_t>(result);
      auto self_info = cuda::detail::getTensorInfo<scalar_t, int32_t>(self);
      triu_tril_kernel<scalar_t, int32_t, upper>
        <<<dim_grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(
          result_info, self_info, k, N);
    } else {
      auto result_info = cuda::detail::getTensorInfo<scalar_t, int64_t>(result);
      auto self_info = cuda::detail::getTensorInfo<scalar_t, int64_t>(self);
      triu_tril_kernel<scalar_t, int64_t, upper>
        <<<dim_grid, dim_block, 0, at::cuda::getCurrentCUDAStream()>>>(
          result_info, self_info, k, N);
    }
  });
  AT_CUDA_CHECK(hipGetLastError());
  return result;
}

Tensor& tril_cuda_(Tensor &self, int64_t k) {
  return tril_cuda_out(self, self, k);
}

Tensor& tril_cuda_out(Tensor &result, const Tensor& self, int64_t k) {
  if (result.sizes() != self.sizes()) {
    result.resize_as_(self);
  }
  if (self.numel() == 0) {
    return result;
  }
  return triu_tril_cuda_template<false>(result, self, k, "tril");
}

Tensor& triu_cuda_(Tensor &self, int64_t k) {
  return triu_cuda_out(self, self, k);
}

Tensor& triu_cuda_out(Tensor &result, const Tensor& self, int64_t k) {
  if (result.sizes() != self.sizes()) {
    result.resize_as_(self);
  }
  if (self.numel() == 0) {
    return result;
  }
  return triu_tril_cuda_template<true>(result, self, k, "triu");
}

}  // namespace native
}  // namespace at
