
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THC/generic/THCTensorMathReduce.cu"
#else

accreal THCTensor_(sumall)(THCState *state, THCTensor *self) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  accreal val;
  if (!THC_reduceAll<scalar_t>(state, self,
                           thrust::identity<accreal>{},
                           ReduceAdd<accreal>{},
                           scalar_cast<accreal>(0),
                           &val, 0)) {
    THArgCheck(false, 1, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
  return val;
}

void THCTensor_(max)(THCState *state,
                     THCTensor *values,
                     THCudaLongTensor *indices,
                     THCTensor *src,
                     int dimension,
                     int keepdim) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, values, indices, src));

  thrust::pair<scalar_t, int64_t>
    init =
    thrust::make_pair<scalar_t, int64_t>(
      THCNumerics<scalar_t>::lower_bound(), 0);

  return THC_reduceDimIndex<scalar_t, int64_t>(
    state, values, indices, src, dimension, keepdim, init,
    MaxValuePair<scalar_t, int64_t>());
}

void THCTensor_(min)(THCState *state,
                     THCTensor *values,
                     THCudaLongTensor *indices,
                     THCTensor *src,
                     int dimension,
                     int keepdim) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, values, indices, src));

  thrust::pair<scalar_t, int64_t>
    init =
    thrust::make_pair<scalar_t, int64_t>(
      THCNumerics<scalar_t>::upper_bound(), 0);

  return THC_reduceDimIndex<scalar_t, int64_t>(
    state, values, indices, src, dimension, keepdim, init,
    MinValuePair<scalar_t, int64_t>());
}

scalar_t THCTensor_(minall)(THCState *state, THCTensor *self) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  THArgCheck(
      THTensor_(nElement)(self) > 0,
      1,
      "cannot perform reduction function min "
      "on tensor with no elements because the "
      "operation does not have an identity"
  );
  accreal val;
  if (!THC_reduceAll<scalar_t>(state, self,
                           thrust::identity<accreal>{},
                           ReduceMin<accreal>{},
                           THCNumerics<accreal>::upper_bound(), &val, 0)) {
    THArgCheck(false, 1, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
  return scalar_cast<scalar_t>(val);
}

scalar_t THCTensor_(maxall)(THCState *state, THCTensor *self) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  THArgCheck(
      THTensor_(nElement)(self) > 0,
      1,
      "cannot perform reduction function max "
      "on tensor with no elements because the "
      "operation does not have an identity"
  );
  accreal val;
  if (!THC_reduceAll<scalar_t>(state, self,
                           thrust::identity<accreal>{},
                           ReduceMax<accreal>{},
                           THCNumerics<accreal>::lower_bound(), &val, 0)) {
    THArgCheck(false, 1, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
  return scalar_cast<scalar_t>(val);
}

#if !defined(THC_REAL_IS_BOOL)

void THCTensor_(prod)(THCState* state, THCTensor *self, THCTensor *src, int dimension, int keepdim) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));
  if (!THC_reduceDim<scalar_t>(state, self, src,
                           thrust::identity<accreal>{},
                           ReduceMultiply<accreal>{},
                           thrust::identity<accreal>{},
                           scalar_cast<accreal>(1),
                           dimension,
                           keepdim)) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(cudaGetLastError());
}

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)

void THCTensor_(renorm)(THCState *state, THCTensor* self, THCTensor* src, scalar_t value, int dimension, scalar_t maxnorm)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));
  dimension = at::maybe_wrap_dim(dimension, src);
  THArgCheck(dimension >= 0 && dimension < THCTensor_(nDimensionLegacyNoScalars)(state, src), 3, "invalid dimension");
  THArgCheck(THCNumerics<scalar_t>::gt(value, scalar_cast<scalar_t>(0)), 2, "non-positive-norm not supported");
  THArgCheck(THCTensor_(nDimensionLegacyNoScalars)(state, src) > 1, 1, "need at least 2 dimensions");

  THCTensor *self_;
  THCTensor *src_ = THCTensor_(newTranspose)(state, src, dimension, 0);
  THCTensor *data = THCTensor_(newClone)(state, src_);
  int64_t numel = THCTensor_(nElement)(state, data);

  if (numel > 0) {
    ptrdiff_t size = numel / THTensor_sizeLegacyNoScalars(data, 0);
    dim3 grid( THTensor_sizeLegacyNoScalars(data, 0));
    // NOTE: only with this specific number of threads can this work on GPUs with a warp size != 32 (such as AMD). Do not alter w/o changing buffer size in kernel.
    dim3 threads(32);

    THCTensor_kernel_renorm<scalar_t, accreal>
      <<<grid, threads, 0, c10::cuda::getCurrentCUDAStream()>>>
      (THCTensor_(data)(state, data), scalar_cast<accreal>(value), size, scalar_cast<accreal>(maxnorm));

    cudaError_t errcode = cudaGetLastError();
    if(errcode != cudaSuccess)
      THError(cudaGetErrorString(errcode));
  }

  THCTensor_(free)(state, src_);
  self_ = THCTensor_(newTranspose)(state, data, dimension, 0);
  THCTensor_(resizeAs)(state, self, self_);
  THCTensor_(freeCopyTo)(state, self_, self);
  THCTensor_(free)(state, data);
}

accreal THCTensor_(std_all)(THCState *state, THCTensor *self, bool unbiased)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  return THCNumerics<accreal>::sqrt((THCTensor_(var_all)(state, self, unbiased)));
}

accreal THCTensor_(var_all)(THCState *state, THCTensor *self, bool unbiased)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  accreal mean = THCTensor_(meanall)(state, self);

  accreal val;
  if (!THC_reduceAll<scalar_t>(state, self,
                           SquareFunctor<accreal>(mean),
                           ReduceAdd<accreal>(),
                           scalar_cast<accreal>(0),
                           &val, 0)) {
    THArgCheck(false, 1, CUTORCH_DIM_WARNING);
  }

  val = THCNumerics<accreal>::div(
    val,
    scalar_cast<accreal>(std::max<int64_t>(0, THCTensor_(nElement)(state, self) - (unbiased ? 1 : 0)))
  );

  THCudaCheck(cudaGetLastError());
  return val;
}

#endif

accreal THCTensor_(meanall)(THCState *state, THCTensor *self)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self));
  return THCTensor_(sumall)(state, self)/THCTensor_(nElement)(state, self);
}


#endif

#endif
