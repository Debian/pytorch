#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/BinaryOps.h>


// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at { namespace native {

void atan2_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "atan2_cuda", [&]() {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return ::atan2(a, b);
    });
  });
}

void smooth_l1_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "smooth_l1_cuda", [&]() {
    gpu_kernel(iter, [] GPU_LAMBDA (scalar_t a, scalar_t b) -> scalar_t {
      auto z = ::abs(a - b);
      return z < scalar_t(1.) ? scalar_t(0.5) * z * z : z - scalar_t(0.5);
    });
  });
}


void mse_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "mse_cuda", [&]() {
    AT_SKIP_BFLOAT16_IF_NOT_ROCM(scalar_t, "mse_cuda", [&] {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        auto diff = a - b;
        return diff * diff;
      });
    });
  });
}

REGISTER_DISPATCH(atan2_stub, &atan2_kernel_cuda);
REGISTER_DISPATCH(smooth_l1_stub, &smooth_l1_kernel_cuda);
REGISTER_DISPATCH(mse_stub, &mse_kernel_cuda);

}} // namespace at::native
