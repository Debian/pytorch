#include <ATen/Context.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/Dispatch.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/PinnedMemoryAllocator.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/CUDASolver.h>
#include <ATen/cuda/CUDABlas.h>
#include <ATen/cuda/CUDAEvent.h>
#include <c10/cuda/CUDAStream.h>

#include <ATen/native/LinearAlgebraUtils.h>
#include <ATen/native/cuda/MiscUtils.h>
#include <ATen/native/cuda/BatchLinearAlgebraLib.h>

#ifdef USE_CUSOLVER

namespace at {
namespace native {

inline static Tensor column_major_identity_matrix_like(const Tensor& self) {
  auto size = self.sizes();
  auto size_slice = IntArrayRef(size.data(), size.size()-1);
  return at::ones(size_slice, self.options()).diag_embed().transpose(-2, -1);
}

template <typename scalar_t>
inline static void _apply_single_inverse_helper(scalar_t* self_ptr, scalar_t* self_inv_ptr, int* ipiv_ptr, int* info_ptr, int n) {
  // self_inv_ptr should already be an identity matrix

  auto handle = at::cuda::getCurrentCUDASolverDnHandle();
  at::cuda::solver::getrf<scalar_t>(handle, n, n, self_ptr, n, ipiv_ptr, info_ptr);
  at::cuda::solver::getrs<scalar_t>(handle, n, n, self_ptr, n, ipiv_ptr, self_inv_ptr, n, info_ptr);
}

template <typename scalar_t>
static void apply_batched_inverse_lib(Tensor& self, Tensor& self_inv, Tensor& infos) {
  const int batch_size = cuda_int_cast(batchCount(self), "batchCount");
  const int n = cuda_int_cast(self.size(-2), "self.size(-2)");

  auto self_data = self.data_ptr<scalar_t>();
  auto self_mat_stride = matrixStride(self);
  auto self_inv_data = self_inv.data_ptr<scalar_t>();
  auto self_inv_mat_stride = matrixStride(self_inv);

  auto& allocator = *::c10::cuda::CUDACachingAllocator::get();

  if (use_loop_launch(batch_size, n)) {
    int* p_infos = infos.data_ptr<int>();
    auto main_stream = at::cuda::getCurrentCUDAStream();

    for (int64_t i = 0; i < batch_size; i++) {
      auto stream = at::cuda::getStreamFromPool();
      at::cuda::CUDAStreamGuard guard(stream);

      at::cuda::CUDAEvent can_start;
      can_start.record(main_stream);
      can_start.block(main_stream);

      int* pivot = reinterpret_cast<int*>(allocator.allocate(sizeof(int) * n).get());
      _apply_single_inverse_helper<scalar_t>(
        &self_data[i * self_mat_stride], &self_inv_data[i * self_inv_mat_stride], pivot, p_infos + i, n);

      at::cuda::CUDAEvent finished;
      finished.record(stream);
      finished.block(main_stream);
    }
  } else {
    // cublas batched kernels require input be "device array of device pointers"
    Tensor self_array = at::arange(
      reinterpret_cast<long>(self_data),
      reinterpret_cast<long>(&self_data[(batch_size-1) * self_mat_stride]) + 1,
      static_cast<long>(self_mat_stride * sizeof(scalar_t)), self.options().dtype(at::kLong));
    Tensor self_inv_array = at::arange(
      reinterpret_cast<long>(self_inv_data),
      reinterpret_cast<long>(&self_inv_data[(batch_size-1) * self_inv_mat_stride]) + 1,
      static_cast<long>(self_inv_mat_stride * sizeof(scalar_t)), self.options().dtype(at::kLong));

    int* ipiv_array = reinterpret_cast<int*>(allocator.allocate(sizeof(int)*batch_size*n).get());

    at::cuda::blas::getrfBatched<scalar_t>(n, reinterpret_cast<scalar_t**>(self_array.data_ptr()), n,
      ipiv_array, infos.data_ptr<int>(), batch_size);

    at::cuda::blas::getriBatched<scalar_t>(n, reinterpret_cast<scalar_t**>(self_array.data_ptr()), n,
      ipiv_array, infos.data_ptr<int>(), batch_size, reinterpret_cast<scalar_t**>(self_inv_array.data_ptr()));
  }
}

template <typename scalar_t>
static void apply_single_inverse_lib(const Tensor& self, Tensor& self_inv, int64_t& info) {
  int n = cuda_int_cast(self.size(-2), "self.size(-2)");

  Tensor ipiv = at::empty({n}, self.options().dtype(at::kInt));
  Tensor info_tmp = at::zeros({1}, self.options().dtype(at::kInt));

  _apply_single_inverse_helper<scalar_t>(
    self.data_ptr<scalar_t>(), self_inv.data_ptr<scalar_t>(), ipiv.data_ptr<int>(), info_tmp.data_ptr<int>(), n);

  info = info_tmp.item<int>();
}

Tensor _inverse_helper_cuda_lib(const Tensor& self) {
  Tensor self_working_copy = cloneBatchedColumnMajor(self);
  Tensor self_inv_working_copy = column_major_identity_matrix_like(self_working_copy);
  const int batch_size = cuda_int_cast(batchCount(self), "batchCount");

  if (self.dim() > 2 && batch_size > 1) {
    Tensor infos = at::zeros({batchCount(self)}, self.options().dtype(kInt));
    AT_DISPATCH_FLOATING_TYPES(self.scalar_type(), "inverse_cuda", [&]{
      apply_batched_inverse_lib<scalar_t>(
        self_working_copy, self_inv_working_copy, infos);
    });
    batchCheckErrors(infos, "inverse_cuda");
  } else {
    int64_t info = 0;
    AT_DISPATCH_FLOATING_TYPES(self.scalar_type(), "inverse_cuda", [&]{
      apply_single_inverse_lib<scalar_t>(self_working_copy, self_inv_working_copy, info);
    });
    singleCheckErrors(info, "inverse_cuda");
  }

  return self_inv_working_copy;
}

}} // namespace at::native

#endif  // USE_CUSOLVER
