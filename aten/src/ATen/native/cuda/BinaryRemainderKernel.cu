#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/BinaryOps.h>

// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at { namespace native {

void remainder_kernel_cuda(TensorIterator& iter) {
  if (isIntegralType(iter.dtype(), /*includeBool*/ false)) {
    AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "remainder_cuda", [&]() {
      gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        scalar_t r = a % b;
        if ((r != 0) && ((r < 0) != (b < 0))) {
          r += b;
        }
        return r;
      });
    });
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "remainder_cuda", [&]() {
      gpu_kernel_with_scalars(iter,
        []GPU_LAMBDA(scalar_t a, scalar_t b) __ubsan_ignore_float_divide_by_zero__ -> scalar_t {
          auto mod = ::fmod(a, b);
          if ((mod != 0) && ((b < 0) != (mod < 0))) mod += b;
          return mod;
        });
    });
  }
}

REGISTER_DISPATCH(remainder_stub, &remainder_kernel_cuda);

}} // namespace at::native
