#include "hip/hip_runtime.h"
#include <limits>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Math.cuh>

namespace at { namespace native {

void bitwise_not_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == ScalarType::Bool) {
    gpu_kernel(iter, []GPU_LAMBDA(bool a) {
      return !a;
    });
  } else {
    AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "bitwise_not_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        return ~a;
      });
    });
  }
}

void exp_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "exp_cuda", [&]() {
    AT_SKIP_BFLOAT16_IF_NOT_ROCM(scalar_t, "exp_cuda", [&] {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        return ::exp(a);
      });
    });
  });
}

void expm1_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "expm1_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::expm1(a);
    });
  });
}

// We manually overload rsqrt because std::rsqrt does not work with complex types.
template<typename scalar_t>
__host__ __device__ static inline scalar_t rsqrt_wrapper(scalar_t v) {
  return ::rsqrt(v);
}

template<typename T>
__host__ __device__ static inline c10::complex<T> rsqrt_wrapper(c10::complex<T> v) {
  const c10::complex<T> one = c10::complex<T>(1.0, 0);
  // std::sqrt for c10::complex is overloaded in c10/util/complex_math.h
  return one / ::sqrt(v);
}

void rsqrt_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(ScalarType::Half, iter.dtype(), "rsqrt_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      // In CUDA, ::rsqrt is overloaded for float and at::Half here is implicitly cast to float.
      return rsqrt_wrapper(a);
    });
  });
}

void sqrt_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.dtype(), "sqrt_cuda", [&]() {
    AT_SKIP_BFLOAT16_IF_NOT_ROCM(scalar_t, "sqrt_cuda", [&] {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        return ::sqrt(a);
      });
    });
  });
}

void sigmoid_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "sigmoid_cuda", [&]() {
    AT_SKIP_BFLOAT16_IF_NOT_ROCM(scalar_t, "sigmoid_cuda", [&] {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        scalar_t one = scalar_t(1);
        return  one / (one + std::exp(- a));
      });
    });
  });
}

void erf_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "erf_cuda", [&]() {
    AT_SKIP_BFLOAT16_IF_NOT_ROCM(scalar_t, "erf_cuda", [&] {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        return ::erf(a);
      });
    });
  });
}

void erfc_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "erfc_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::erfc(a);
    });
  });
}

void erfinv_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "erfinv_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::erfinv(a);
    });
  });
}

void clamp_kernel_cuda(TensorIterator& iter, Scalar min_value, Scalar max_value) {
  AT_DISPATCH_ALL_TYPES_AND(kHalf, iter.dtype(), "clamp_cuda", [&]() {
    auto lower = min_value.to<scalar_t>();
    auto upper = max_value.to<scalar_t>();
    gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t v) -> scalar_t {
      return (v < lower) ? lower : (v > upper ? upper : v);
    });
  });
}

void clamp_min_kernel_cuda(TensorIterator& iter, Scalar min_value) {
  AT_DISPATCH_ALL_TYPES_AND(kHalf, iter.dtype(), "clamp_min_cuda", [&]() {
    auto lower = min_value.to<scalar_t>();
    gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t v) -> scalar_t {
      return v < lower ? lower : v;
    });
  });
}

void clamp_max_kernel_cuda(TensorIterator& iter, Scalar max_value) {
  AT_DISPATCH_ALL_TYPES_AND(kHalf, iter.dtype(), "clamp_max_cuda", [&]() {
    auto upper = max_value.to<scalar_t>();
    gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t v) -> scalar_t {
      return v > upper ? upper : v;
    });
  });
}

REGISTER_DISPATCH(bitwise_not_stub, &bitwise_not_kernel_cuda);
REGISTER_DISPATCH(exp_stub, &exp_kernel_cuda);
REGISTER_DISPATCH(expm1_stub, &expm1_kernel_cuda);
REGISTER_DISPATCH(rsqrt_stub, &rsqrt_kernel_cuda);
REGISTER_DISPATCH(sqrt_stub, &sqrt_kernel_cuda);
REGISTER_DISPATCH(sigmoid_stub, &sigmoid_kernel_cuda);
REGISTER_DISPATCH(erf_stub, &erf_kernel_cuda);
REGISTER_DISPATCH(erfc_stub, &erfc_kernel_cuda);
REGISTER_DISPATCH(erfinv_stub, &erfinv_kernel_cuda);
REGISTER_DISPATCH(clamp_stub, &clamp_kernel_cuda);
REGISTER_DISPATCH(clamp_min_stub, &clamp_min_kernel_cuda);
REGISTER_DISPATCH(clamp_max_stub, &clamp_max_kernel_cuda);
}}
