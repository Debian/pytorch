#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/LegacyTHFunctionsCUDA.h>
#include <ATen/NamedTensorUtils.h>
#include <ATen/cuda/CUDABlas.h>

namespace at { namespace native {

Tensor baddbmm_cuda(const Tensor& self, const Tensor& batch1, const Tensor& batch2, Scalar beta, Scalar alpha) {
  Tensor b_self;
  std::tie(b_self) = expand_size(self, {batch1.size(0), batch1.size(1), batch2.size(2)}, "baddbmm");
  return legacy::cuda::_th_baddbmm(b_self, batch1, batch2, beta, alpha);
}

Tensor& baddbmm_out_cuda(Tensor &result, const Tensor& self, const Tensor& batch1, const Tensor& batch2, Scalar beta, Scalar alpha) {
  Tensor b_self;
  std::tie(b_self) = expand_size(self, {batch1.size(0), batch1.size(1), batch2.size(2)}, "baddbmm_out");
  return legacy::cuda::_th_baddbmm_out(result, b_self, batch1, batch2, beta, alpha);
}

Tensor& baddbmm__cuda(Tensor& self, const Tensor& batch1, const Tensor& batch2, Scalar beta, Scalar alpha) {
  return baddbmm_out_cuda(self, self, batch1, batch2, beta, alpha);
}

Tensor& bmm_out_cuda(Tensor &result, const Tensor& batch1, const Tensor& batch2) {
  result.resize_({ batch1.size(0), batch1.size(1), batch2.size(2) });
  return legacy::cuda::_th_bmm_out(result, batch1, batch2);
}

Tensor bmm_cuda(const Tensor& self, const Tensor& mat2) {
  Tensor result = at::empty({0}, self.options());
  return native::bmm_out_cuda(result, self, mat2);
}

Tensor prepare_matrix_for_cublas(Tensor& tensor, bool& transpose_tensor) {
  Tensor tensor_;
  IntArrayRef tensor_strides = tensor.strides();
  IntArrayRef tensor_sizes = tensor.sizes();

  if ((tensor_strides[0] == 1) && (tensor_strides[1] >= std::max<int64_t>(1, tensor_sizes[0]))) {
    tensor_ = tensor;
    transpose_tensor = false;
  } else if ((tensor_strides[1] == 1) && (tensor_strides[0] >= std::max<int64_t>(1, tensor_sizes[1]))) {
    tensor_ = tensor;
    transpose_tensor = true;
  } else {
    transpose_tensor = true;
    tensor_ = tensor.clone(at::MemoryFormat::Contiguous);
  }

  return tensor_;
}

namespace {

Tensor& addmm_out_cuda_impl(Tensor& result, const Tensor& self, const Tensor& mat1, const Tensor& mat2, Scalar beta, Scalar alpha) {
  TORCH_CHECK(mat1.dim() == 2 && mat2.dim() == 2, "tensors must be 2-D");

  TensorArg args[]{{result, "out", 0}, {self, "self", 1}, {mat1, "mat1", 2}, {mat2, "mat2", 3}};
  checkAllSameGPU("addmm", args);

  Tensor self_;
  if (&result != &self) {
    std::tie(self_) = expand_size(self, {mat1.size(0), mat2.size(1)}, "addmm");
  } else {
    self_ = self;
  }

  IntArrayRef mat1_sizes = mat1.sizes();
  IntArrayRef mat2_sizes = mat2.sizes();
  IntArrayRef self__sizes = self_.sizes();
  TORCH_CHECK(mat1_sizes[1] == mat2_sizes[0], "mat1 dim 1 must match mat2 dim 0");
  TORCH_CHECK(self__sizes[0] == mat1_sizes[0], "self_ dim 0 must match mat1 dim 0");
  TORCH_CHECK(self__sizes[1] == mat2_sizes[1], "self_ dim 1 must match mat2 dim 1");

  if (&result != &self) {
    at::native::resize_as_(result, self_);
    if (beta.toComplexDouble() != 0.0) {
      at::native::copy_(result, self_);
    }
  }

  TORCH_CHECK(result.dim() == 2 && self_.dim() == 2, "tensors must be 2-D");

  IntArrayRef result_sizes = result.sizes();
  if ((result_sizes[0] == 0) || (result_sizes[1] == 0)) {
    return result;
  }

  bool transpose_result;
  Tensor result_ = prepare_matrix_for_cublas(result, transpose_result);
  bool transpose_mat1;
  bool transpose_mat2;
  Tensor mat1_ = transpose_result ? mat2 : mat1;
  Tensor mat2_ = transpose_result ? mat1 : mat2;
  mat1_ = prepare_matrix_for_cublas(mat1_, transpose_mat1);
  mat2_ = prepare_matrix_for_cublas(mat2_, transpose_mat2);

  if (transpose_result) {
    transpose_mat1 = !transpose_mat1;
    transpose_mat2 = !transpose_mat2;
    mat1_sizes = mat1_.sizes();
    mat2_sizes = mat2_.sizes();
  }

  int64_t m = mat1_sizes[transpose_result ? 1 : 0];
  int64_t k = mat1_sizes[transpose_result ? 0 : 1];
  int64_t n = mat2_sizes[transpose_result ? 0 : 1];
  int64_t mat1_ld = mat1_.stride((transpose_mat1 == transpose_result) ? 1 : 0);
  int64_t mat2_ld = mat2_.stride((transpose_mat2 == transpose_result) ? 1 : 0);
  int64_t result_ld = result_.stride(transpose_result ? 0 : 1);
  at::ScalarType scalar_type = self_.scalar_type();

  if (mat1.numel() == 0) {
    // By definition, when beta==0, values in self should be ignored. nans and infs
    // should not propagate
    if (beta.toComplexDouble() == 0.) {
      return result.zero_();
    }
    return at::native::mul_out(result, self, at::native::scalar_tensor(beta, at::device(at::kCPU).dtype(self.scalar_type())));
  }

  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, scalar_type, "addmm_cuda", [&] {
    scalar_t alpha_val = alpha.to<scalar_t>();
    scalar_t beta_val = beta.to<scalar_t>();
    scalar_t* mat1_ptr = mat1_.data_ptr<scalar_t>();
    scalar_t* mat2_ptr = mat2_.data_ptr<scalar_t>();
    scalar_t* result_ptr = result_.data_ptr<scalar_t>();
    at::cuda::blas::gemm<scalar_t>(
      transpose_mat1 ? 't' : 'n',
      transpose_mat2 ? 't' : 'n',
      m, n, k,
      alpha_val,
      mat1_ptr, mat1_ld,
      mat2_ptr, mat2_ld,
      beta_val,
      result_ptr, result_ld
    );
  });
  if (result.data_ptr() != result_.data_ptr()) {
    result.copy_(result_);
  }
  return result;
}

} // anonymous namespace

Tensor& mm_out_cuda(Tensor& result, const Tensor& self, const Tensor& mat2) {
  result.resize_({ self.size(0), mat2.size(1) });
  return addmm_out_cuda_impl(result, result, self, mat2, 0, 1);
}

Tensor mm_cuda(const Tensor& self, const Tensor& mat2) {
  Tensor result = at::empty({ self.size(0), mat2.size(1) }, self.options());
  return addmm_out_cuda_impl(result, result, self, mat2, 0, 1);
}

Tensor& addmm_out_cuda(Tensor &out, const Tensor &self,
                        const Tensor &mat1, const Tensor &mat2,
                        Scalar beta, Scalar alpha) {
  {
    at::NoNamesGuard guard;
    Tensor& result = addmm_out_cuda_impl(out, self, mat1, mat2, beta, alpha);
  }
  at::namedinference::propagate_names_for_addmm(out, mat1, mat2, self);
  return out;
}

Tensor addmm_cuda(const Tensor& self, const Tensor& mat1, const Tensor& mat2,
                  Scalar beta, Scalar alpha) {
  Tensor out = at::empty({0}, self.options());
  addmm_out_cuda(out, self, mat1, mat2, beta, alpha);
  return out;
}

Tensor& addmm__cuda(Tensor& self, const Tensor& mat1, const Tensor& mat2,
                    Scalar beta, Scalar alpha) {
  addmm_out_cuda(self, self, mat1, mat2, beta, alpha);
  return self;
}

Tensor& addbmm_out_cuda(Tensor& out, const Tensor& self,
                        const Tensor& batch1, const Tensor& batch2,
                        Scalar beta, Scalar alpha) {
  TORCH_CHECK(batch1.dim() == 3 && batch2.dim() == 3,
              "Batch tensors should be 3D, got dimensions ", batch1.dim(),
              " and ", batch2.dim());

  Tensor self_;
  if (&out != &self) {
    std::tie(self_) = expand_size(self, {batch1.size(1), batch2.size(2)}, "addbmm");
  } else {
    self_ = self;
  }

  TORCH_CHECK(out.device() == self_.device() &&
              out.device() == batch1.device() &&
              out.device() == batch2.device(),
              "Expected all tensors to be on the same device. Found: ",
              out.device(), ", ", self_.device(), ", ",
              batch1.device(), " and ", batch2.device());
  TORCH_CHECK(self_.dim() == 2,
              "2D tensor expected, got ", self_.dim(), "D tensor for input");
  int64_t batchnum = batch1.size(0);
  int64_t m1d1 = batch1.size(1);
  int64_t innerdim = batch1.size(2);
  int64_t m2d2 = batch2.size(2);
  TORCH_CHECK(batchnum == batch2.size(0),
              "equal number of batches expected");
  TORCH_CHECK(m1d1 == self_.size(0),
              "first dimension of batch1  must match first dimension of input");
  TORCH_CHECK(m2d2 == self_.size(1),
              "second dimension of batch2 must match second dimension of input");
  TORCH_CHECK(innerdim == batch2.size(1),
              "second dimension of batch1 must match first dimension of batch2");

  if (&out != &self) {
    at::native::resize_as_(out, self_);
    if (beta.to<double>() != 0.0) {
      at::native::copy_(out, self_);
    }
  }

  for (int64_t i=0; i<batchnum; i++) {
    addmm_out_cuda(out, out, batch1[i], batch2[i], beta, alpha);
    beta = 1;
  }
  return out;
}

Tensor& addbmm__cuda(Tensor& self,
                     const Tensor& batch1, const Tensor& batch2,
                     Scalar beta, Scalar alpha) {
  addbmm_out_cuda(self, self, batch1, batch2, beta, alpha);
  return self;
}

Tensor addbmm_cuda(const Tensor& self,
                   const Tensor& batch1, const Tensor& batch2,
                   Scalar beta, Scalar alpha)
{
  Tensor out = at::empty({0}, self.options());
  addbmm_out_cuda(out, self, batch1, batch2, beta, alpha);
  return out;
}

namespace {

inline void dot_check(const Tensor& self, const Tensor& other) {
  TORCH_CHECK(
      self.dim() == 1 && other.dim() == 1,
      "1D tensors expected, but got ",
      self.dim(),
      "D and ",
      other.dim(),
      "D tensors");
  TORCH_CHECK(
      self.scalar_type() == other.scalar_type(),
      "dot : expected both vectors to have same dtype, but found ",
      self.scalar_type(),
      " and ",
      other.scalar_type());
  TORCH_CHECK(
      self.numel() == other.numel(),
      "inconsistent tensor size, expected tensor [",
      self.numel(),
      "] and src [",
      other.numel(),
      "] to have the same number of elements, but got ",
      self.numel(),
      " and ",
      other.numel(),
      " elements respectively");
  TORCH_CHECK(
      self.device() == other.device(),
      "expected all tensors to be on the same device. Found: ",
      self.device(),
      ", ",
      other.device());
  TORCH_CHECK(
      (self.numel() <= INT_MAX) && (self.stride(0) <= INT_MAX) &&
          (other.stride(0) <= INT_MAX),
      "dot only supports n, incx, incy with the bound [val] <= %d",
      INT_MAX);
}

} // anonymous namespace

Tensor dot_cuda(const Tensor& self, const Tensor& other) {
  at::NoNamesGuard guard;

  dot_check(self, other);

  const int n = static_cast<int>(self.numel());
  int incx = static_cast<int>(self.stride(0));
  int incy = static_cast<int>(other.stride(0));
  if (n == 1) {
    incx = 1;
    incy = 1;
  }

  return AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(ScalarType::Half, self.scalar_type(), "dot", [&] {
    Tensor result = at::empty({}, self.options());

    auto handle = at::cuda::getCurrentCUDABlasHandle();
    at::cuda::blas::PointerModeGuard pointerModeGuard(handle, HIPBLAS_POINTER_MODE_DEVICE);
    at::cuda::blas::dot<scalar_t>(
        handle,
        n,
        self.data_ptr<scalar_t>(),
        incx,
        other.data_ptr<scalar_t>(),
        incy,
        result.data_ptr<scalar_t>());

    return result;
  });
}

Tensor vdot_cuda(const Tensor& self, const Tensor& other) {
  if (!self.is_complex()) {
    return dot_cuda(self, other);
  }

  at::NoNamesGuard guard;
  dot_check(self, other);

  const int n = static_cast<int>(self.numel());
  int incx = static_cast<int>(self.stride(0));
  int incy = static_cast<int>(other.stride(0));
  if (n == 1) {
    incx = 1;
    incy = 1;
  }

  return AT_DISPATCH_COMPLEX_TYPES(self.scalar_type(), "vdot", [&] {
    Tensor result = at::empty({}, self.options());

    auto handle = at::cuda::getCurrentCUDABlasHandle();
    at::cuda::blas::PointerModeGuard pointerModeGuard(
        handle, HIPBLAS_POINTER_MODE_DEVICE);
    at::cuda::blas::vdot<scalar_t>(
        handle,
        n,
        self.data_ptr<scalar_t>(),
        incx,
        other.data_ptr<scalar_t>(),
        incy,
        result.data_ptr<scalar_t>());

    return result;
  });
}
} }
