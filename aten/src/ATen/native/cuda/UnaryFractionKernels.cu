#include "hip/hip_runtime.h"
#include <limits>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Math.cuh>
#include <ATen/native/cuda/zmath.cuh>

namespace at { namespace native {

// We manually overload ceil because std::ceil does not work with std::complex types.
template <typename scalar_t>
__host__ __device__ static inline scalar_t ceil_wrapper(scalar_t a) {
  return std::ceil(a);
}

template<typename T>
__host__ __device__ static inline std::complex<T> ceil_wrapper(std::complex<T> v) {
  return std::complex<T>(std::ceil(v.real()), std::ceil(v.imag()));
}

void ceil_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(ScalarType::Half, iter.dtype(), "ceil_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ceil_wrapper(a);
    });
  });
}

void frac_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "frac_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return a - ::trunc(a);
    });
  });
}

// We manually overload floor because std::floor does not work with std::complex types.
template <typename scalar_t>
__host__ __device__ static inline scalar_t floor_wrapper(scalar_t a) {
  return std::floor(a);
}

template<typename T>
__host__ __device__ static inline std::complex<T> floor_wrapper(std::complex<T> v) {
  return std::complex<T>(std::floor(v.real()), std::floor(v.imag()));
}

void floor_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(ScalarType::Half, iter.dtype(), "floor_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return floor_wrapper(a);
    });
  });
}

void reciprocal_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "reciprocal_cuda", [&]() {
    using acc_t = acc_type<scalar_t, /*is_cuda=*/true>;
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return static_cast<acc_t>(1) / a;
    });
  });
}

// We manually overload nearbyint because std::nearbyint does not work with std::complex types and ROCm.
template <typename scalar_t>
__host__ __device__ static inline scalar_t nearbyint_wrapper(scalar_t a) {
  return static_cast<scalar_t>(::nearbyintf(static_cast<float>(a)));
}

__host__ __device__ static inline double nearbyint_wrapper(double a) {
  return ::nearbyint(a);
}

__host__ __device__ static inline std::complex<float> nearbyint_wrapper(std::complex<float> a) {
  return std::complex<float>(::nearbyintf(static_cast<float>(a.real())), ::nearbyintf(static_cast<float>(a.imag())));
}

__host__ __device__ static inline std::complex<double> nearbyint_wrapper(std::complex<double> a) {
  return std::complex<double>(::nearbyint(static_cast<double>(a.real())), ::nearbyint(static_cast<double>(a.imag())));
}

void round_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(ScalarType::Half, iter.dtype(), "round_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      // We do not use std::round because we would like to round midway numbers to the nearest even integer.
      return nearbyint_wrapper(a);
    });
  });
}

// We manually overload trunc because std::trunc does not work with std::complex types and ROCm.
template <typename scalar_t>
__host__ __device__ static inline scalar_t trunc_wrapper(scalar_t a) {
  return static_cast<scalar_t>(::truncf(static_cast<float>(a)));
}

__host__ __device__ static inline double trunc_wrapper(double a) {
  return ::trunc(a);
}

__host__ __device__ static inline std::complex<float> trunc_wrapper(std::complex<float> a) {
  return std::complex<float>(::truncf(static_cast<float>(a.real())), ::truncf(static_cast<float>(a.imag())));
}

__host__ __device__ static inline std::complex<double> trunc_wrapper(std::complex<double> a) {
  return std::complex<double>(::trunc(static_cast<double>(a.real())), ::trunc(static_cast<double>(a.imag())));
}

void trunc_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(ScalarType::Half, iter.dtype(), "trunc_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return trunc_wrapper(a);
    });
  });
}

REGISTER_DISPATCH(ceil_stub, &ceil_kernel_cuda);
REGISTER_DISPATCH(frac_stub, &frac_kernel_cuda);
REGISTER_DISPATCH(floor_stub, &floor_kernel_cuda);
REGISTER_DISPATCH(reciprocal_stub, &reciprocal_kernel_cuda);
REGISTER_DISPATCH(round_stub, &round_kernel_cuda);
REGISTER_DISPATCH(trunc_stub, &trunc_kernel_cuda);

}} // namespace at::native
