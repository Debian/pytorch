#include "hip/hip_runtime.h"
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Reduce.cuh>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/SharedReduceOps.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/NumericLimits.cuh>
#include <THC/THCNumerics.cuh>
#include <ATen/native/ReduceOps.h>
#include<ATen/native/ReduceAllOps.h>
#include <ATen/native/ReduceOpsUtils.h>
#include <ATen/native/TensorCompare.h>


namespace at { namespace native {

template <typename scalar_t, typename acc_t=scalar_t>
void max_values_kernel_cuda_impl(TensorIterator& iter) {
  gpu_reduce_kernel<scalar_t, scalar_t>(
    iter, func_wrapper<acc_t> ([]GPU_LAMBDA(acc_t a, acc_t b) -> acc_t {
      return (THCNumerics<acc_t>::isnan(a) || a > b) ? a : b;
    }), at::numeric_limits<acc_t>::lower_bound());
}

template <typename scalar_t, typename acc_t=scalar_t>
void min_values_kernel_cuda_impl(TensorIterator& iter) {
  gpu_reduce_kernel<scalar_t, scalar_t>(
    iter, func_wrapper<acc_t> ([]GPU_LAMBDA(acc_t a, acc_t b) -> acc_t {
      return (THCNumerics<acc_t>::isnan(a) || a < b) ? a : b;
    }), at::numeric_limits<acc_t>::upper_bound());
}

void max_values_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype(1) == kHalf) {
    max_values_kernel_cuda_impl<at::Half, float>(iter);
  } else {
    AT_DISPATCH_ALL_TYPES(iter.dtype(), "max_values_cuda", [&]() {
      max_values_kernel_cuda_impl<scalar_t>(iter);
    });
  }
}

void min_values_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype(1) == kHalf) {
    min_values_kernel_cuda_impl<at::Half, float>(iter);
  } else {
    AT_DISPATCH_ALL_TYPES(iter.dtype(), "min_values_cuda", [&]() {
      min_values_kernel_cuda_impl<scalar_t>(iter);
    });
  }
}

template <typename scalar_t, typename acc_t=scalar_t>
void argmax_kernel_cuda_impl(TensorIterator& iter) {
  gpu_reduce_kernel<scalar_t, int64_t>(
    iter,
    ArgMaxOps<acc_t>{},
    thrust::pair<acc_t, int64_t>(at::numeric_limits<acc_t>::lower_bound(), 0));
};

template <typename scalar_t, typename acc_t=scalar_t>
void argmin_kernel_cuda_impl(TensorIterator& iter) {
  gpu_reduce_kernel<scalar_t, int64_t>(
    iter,
    ArgMinOps<acc_t>{},
    thrust::pair<acc_t, int64_t>(at::numeric_limits<acc_t>::upper_bound(), 0));
};

void argmax_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype(1) == kHalf) {
    // Instead of implementing is_nan and warp_shfl_down
    // we can convert halves to float and do all the operations in float
    argmax_kernel_cuda_impl<at::Half, float>(iter);
  } else {
    AT_DISPATCH_ALL_TYPES(iter.dtype(1), "argmax_cuda", [&]() {
      argmax_kernel_cuda_impl<scalar_t>(iter);
    });
  }
}

void argmin_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype(1) == kHalf) {
    // Instead of implementing is_nan and warp_shfl_down
    // we can convert halves to float and do all the operations in float
    argmin_kernel_cuda_impl<at::Half, float>(iter);
  } else {
    AT_DISPATCH_ALL_TYPES(iter.dtype(1), "argmin_cuda", [&]() {
      argmin_kernel_cuda_impl<scalar_t>(iter);
    });
  }
}

static void min_kernel_impl(Tensor& result, Tensor& indice, const Tensor& self, int64_t dim, bool keepdim) {
  at::TensorIterator iter = make_reduction("min", result, indice, self, dim, keepdim, self.scalar_type(), kLong);
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBool, iter.dtype(2), "min_cuda", [&]() {
    gpu_reduce_kernel<scalar_t, scalar_t>(
      iter,
      MinOps<scalar_t>{},
      thrust::pair<scalar_t, int64_t>(at::numeric_limits<scalar_t>::upper_bound(), 0));
  });
}

static void max_kernel_impl(Tensor& result, Tensor& indice, const Tensor& self, int64_t dim, bool keepdim) {
  at::TensorIterator iter = make_reduction("max", result, indice, self, dim, keepdim, self.scalar_type(), kLong);
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBool, iter.dtype(2), "max_cuda", [&]() {
    gpu_reduce_kernel<scalar_t, scalar_t>(
      iter,
      MaxOps<scalar_t>{},
      thrust::pair<scalar_t, int64_t>(at::numeric_limits<scalar_t>::lower_bound(), 0));
  });
}

static void min_all_kernel_impl(Tensor& result, const Tensor& input) {
  auto dtype = input.scalar_type();
  auto iter = make_reduction("min_all", result, input, std::vector<int64_t>{}, false, dtype);
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBool, dtype, "min_all_cuda", [&] {
    min_values_kernel_cuda_impl<scalar_t>(iter);
  });
}

static void max_all_kernel_impl(Tensor& result, const Tensor& input) {
  auto dtype = input.scalar_type();
  auto iter = make_reduction("min_all", result, input, std::vector<int64_t>{}, false, dtype);
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBool, dtype, "max_all_cuda", [&] {
    max_values_kernel_cuda_impl<scalar_t>(iter);
  });
}

REGISTER_DISPATCH(max_values_stub, &max_values_kernel_cuda);
REGISTER_DISPATCH(min_values_stub, &min_values_kernel_cuda);
REGISTER_DISPATCH(argmax_stub, &argmax_kernel_cuda);
REGISTER_DISPATCH(argmin_stub, &argmin_kernel_cuda);
REGISTER_DISPATCH(min_stub, &min_kernel_impl);
REGISTER_DISPATCH(max_stub, &max_kernel_impl);
REGISTER_DISPATCH(min_all_stub, &min_all_kernel_impl);
REGISTER_DISPATCH(max_all_stub, &max_all_kernel_impl);

}} // namespace at::native
