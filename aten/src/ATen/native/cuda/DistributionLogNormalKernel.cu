#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/ExpandUtils.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/CUDAGenerator.h>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/DistributionTemplates.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>
#include <utility>
#include <functional>

#include <ATen/native/Distributions.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/LegacyTHFunctionsCUDA.h>

#include <THC/THCGeneral.h>
#include <THC/THCApply.cuh>
#include <THC/THCDeviceUtils.cuh>

#include <cstdint>
#include <limits>
#include <utility>
#include <type_traits>

namespace at { namespace native {

void log_normal_kernel(TensorIterator& iter, double mean_, double std_, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "log_normal_cuda", [&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    auto mean = static_cast<accscalar_t>(mean_);
    auto std = static_cast<accscalar_t>(std_);
    if (std::is_same<scalar_t, double>::value) {
      // define lambda for log_normal transformation
      auto log_normal_func = [mean, std] __device__ (accscalar_t rand) {
        return static_cast<scalar_t>(::exp(rand * std + mean));
      };
      distribution_nullary_kernel<scalar_t, accscalar_t, curand4_engine_calls/2>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_normal2_double(state); },
        log_normal_func);
    } else {
      auto log_normal_func = [mean, std] __device__ (accscalar_t rand) {
        // use __expf fast approximation for peak bandwidth
        return static_cast<scalar_t>(__expf(rand * std + mean));
      };
      distribution_nullary_kernel<scalar_t, accscalar_t, curand4_engine_calls>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_normal4(state); },
        log_normal_func);
    }
   });
}

REGISTER_DISPATCH(log_normal_stub, &log_normal_kernel);

}} // namespace at::native
