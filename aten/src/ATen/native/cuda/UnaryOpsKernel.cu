#include "hip/hip_runtime.h"
#include <limits>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Math.cuh>
#include <ATen/native/cuda/zmath.cuh>

namespace at { namespace native {

void bitwise_not_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == ScalarType::Bool) {
    gpu_kernel(iter, []GPU_LAMBDA(bool a) {
      return !a;
    });
  } else {
    AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "bitwise_not_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        return ~a;
      });
    });
  }
}

void expm1_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "expm1_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::expm1(a);
    });
  });
}

// We manually overload rsqrt because std::rsqrt does not work with thrust::complex types.
template<typename scalar_t>
__host__ __device__ static inline scalar_t rsqrt_wrapper(scalar_t v) {
  return ::rsqrt(v);
}

template<typename T>
__host__ __device__ static inline thrust::complex<T> rsqrt_wrapper(thrust::complex<T> v) {
  const thrust::complex<T> one = thrust::complex<T>(1.0, 0);
  return one/thrust::sqrt(v);
}

void rsqrt_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(ScalarType::Half, iter.dtype(), "rsqrt_cuda", [&]() {
    using thrust_t = typename ztype_cuda<scalar_t>::thrust_t;
    gpu_kernel(iter, []GPU_LAMBDA(thrust_t a) -> thrust_t {
      // In CUDA, ::rsqrt is overloaded for float and at::Half here is implicitly cast to float.
      return rsqrt_wrapper(a);
    });
  });
}

// We manually overload sqrt because std::sqrt does not work with thrust::complex types.
template<typename scalar_t>
__host__ __device__ static inline scalar_t sqrt_wrapper(scalar_t v) {
  return ::sqrt(v);
}

template<typename T>
__host__ __device__ static inline thrust::complex<T> sqrt_wrapper(thrust::complex<T> v) {
  return thrust::sqrt(v);
}

void sqrt_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(ScalarType::Half, iter.dtype(), "sqrt_cuda", [&]() {
    using thrust_t = typename ztype_cuda<scalar_t>::thrust_t;
    gpu_kernel(iter, []GPU_LAMBDA(thrust_t a) -> thrust_t {
      return sqrt_wrapper(a);
    });
  });
}

void sigmoid_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "sigmoid_cuda", [&]() {
    AT_SKIP_BFLOAT16_IF_NOT_ROCM(scalar_t, "sigmoid_cuda", [&] {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        scalar_t one = scalar_t(1);
        return  one / (one + std::exp(- a));
      });
    });
  });
}

void erfinv_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "erfinv_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::erfinv(a);
    });
  });
}

REGISTER_DISPATCH(bitwise_not_stub, &bitwise_not_kernel_cuda);
REGISTER_DISPATCH(expm1_stub, &expm1_kernel_cuda);
REGISTER_DISPATCH(rsqrt_stub, &rsqrt_kernel_cuda);
REGISTER_DISPATCH(sqrt_stub, &sqrt_kernel_cuda);
REGISTER_DISPATCH(sigmoid_stub, &sigmoid_kernel_cuda);
REGISTER_DISPATCH(erfinv_stub, &erfinv_kernel_cuda);
}}
