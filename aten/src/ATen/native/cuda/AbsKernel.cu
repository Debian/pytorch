#include "hip/hip_runtime.h"
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>

namespace at { namespace native {

// We manually overload abs because std::abs does not work with thrust::complex types and ROCm.
template<typename scalar_t>
__host__ __device__ static inline scalar_t abs_wrapper(scalar_t v) {
  return ::abs(v);
}

template<typename T>
__host__ __device__ static inline c10::complex<T> abs_wrapper(c10::complex<T> v) {
  return std::abs(v);
}

__host__ __device__ static inline uint8_t abs_wrapper(uint8_t v) {
  return v;
}

__host__ __device__ static inline bool abs_wrapper(bool v) {
  return v;
}

void abs_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(ScalarType::Half, ScalarType::BFloat16, ScalarType::Bool, iter.dtype(), "abs_cuda", [&]() {
    AT_SKIP_BFLOAT16_IF_NOT_ROCM(scalar_t, "abs_cuda", [&] {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        return abs_wrapper(a);
      });
    });
  });
}

REGISTER_DISPATCH(abs_stub, &abs_kernel_cuda);

}} // namespace at::native
