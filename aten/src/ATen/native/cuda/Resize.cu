#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/core/op_registration/op_registration.h>
#include <ATen/native/cuda/Resize.cuh>
#include <ATen/native/ResizeCommon.h>

namespace at {
namespace native {
namespace {

Tensor& resize_cuda_(
    Tensor& self,
    IntArrayRef size,
    c10::optional<MemoryFormat> optional_memory_format) {
  if (self.has_names()) {
    return resize_named_tensor_(self, size, optional_memory_format);
  }
  auto* self_ = self.unsafeGetTensorImpl();
  resize_impl_cuda_(self_, size, /*strides=*/c10::nullopt);
  if (optional_memory_format.has_value()) {
    auto memory_format =
        optional_memory_format.value();
    TORCH_CHECK(
        memory_format != MemoryFormat::Preserve,
        "Unsupported memory format",
        memory_format);
    self_->empty_tensor_restride(memory_format);
  }
  return self;
}
static auto registry = torch::RegisterOperators()
  .op(torch::RegisterOperators::options()
    .schema("aten::resize_(Tensor(a!) self, int[] size, *, MemoryFormat? memory_format=None) -> Tensor(a!)")
    .impl_unboxedOnlyKernel<decltype(resize_cuda_), &resize_cuda_>(DispatchKey::CUDATensorId))
  ;

} // namespace
} // namespace native
} // namespace at
