#include "hip/hip_runtime.h"
#include <limits>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Math.cuh>

namespace at { namespace native {

void digamma_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "digamma_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return calc_digamma(a);
    });
  });
}

void trigamma_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "trigamma_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return calc_trigamma(a);
    });
  });
}

void polygamma_kernel_cuda(TensorIterator& iter, int64_t n) {
  switch (n) {
    case 0: digamma_kernel_cuda(iter); break;
    case 1: trigamma_kernel_cuda(iter); break;
    default: TORCH_CHECK(false, "polygamma(n,x) is not implemented for n>=2, but was ", n);
  }
}

void lgamma_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "lgamma_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::lgamma(a);
    });
  });
}

REGISTER_DISPATCH(digamma_stub, &digamma_kernel_cuda);
REGISTER_DISPATCH(polygamma_stub, &polygamma_kernel_cuda);
REGISTER_DISPATCH(lgamma_stub, &lgamma_kernel_cuda);

}} // namespace at::native
