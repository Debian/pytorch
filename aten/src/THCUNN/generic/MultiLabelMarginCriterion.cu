
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THCUNN/generic/MultiLabelMarginCriterion.cu"
#else

static inline void THNN_(MultiLabelMarginCriterion_shapeCheck)(
                         THCState *state,
                         THCTensor *input, THCTensor *target) {
  if (input->dim() <= 1) {
    int dim = input->dim() == 0 ? 1 : input->size(0);
    int target_size = target->dim() == 0 ? 1 : target->size(0);
    TORCH_CHECK(!target->is_empty() && (target->dim() <= 1) && (target_size == dim),
                "inconsistent target size: ", target->sizes(), " for input of size: ", input->sizes());
  } else if (input->dim() == 2) {
    int nframe = input->size(0);
    int dim = input->size(1);
    TORCH_CHECK(!target->is_empty() && (target->dim() == 2)
                && (target->size(0) == nframe) && (target->size(1) == dim),
                "inconsistent target size: ", target->sizes(), " for input of size: ", input->sizes());
  } else {
    TORCH_CHECK(false, "non-empty vector or matrix expected, got size: ", input->sizes());
  }
}

// TODO: improve error messages
void THNN_(MultiLabelMarginCriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCIndexTensor *target,
           THCTensor *output,
           THCTensor *istarget,
           int64_t reduction)
{
  #if defined(THC_REAL_IS_BFLOAT16) && !defined(__HIP_PLATFORM_HCC__)
  TORCH_CHECK(false, "MultiLabelMarginCriterion_updateOutput not suppported with BFloat16");
  #else
  THNN_(MultiLabelMarginCriterion_shapeCheck)(state, input, target);
  input = THCTensor_(newContiguous)(state, input);
  target = THCIndexTensor_(newContiguous)(state, target);
  istarget = THCTensor_(newContiguous)(state, istarget);
  THCTensor_(resizeAs)(state, istarget, target);

  if(input->dim() <= 1)
  {
    int dim = input->dim() == 0 ? 1 : input->size(0);
    int target_size = target->dim() == 0 ? 1 : target->size(0);
    THCTensor_(resize0d)(state, output);

    dim3 blocks(1);
    dim3 threads(MULTILABELMARGIN_THREADS);

    cunn_MultiLabelMarginCriterion_updateOutput_kernel<scalar_t, accreal>
      <<<blocks, threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
        THCTensor_(data)(state, output),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        THCTensor_(data)(state, istarget),
        1, dim,
        reduction == at::Reduction::Mean
        );
    THCudaCheck(cudaGetLastError());
  }
  else if(input->dim() == 2)
  {
    int nframe = input->size(0);
    int dim = input->size(1);
    dim3 blocks(input->size(0));
    dim3 threads(MULTILABELMARGIN_THREADS);

    if (reduction != at::Reduction::None)
    {
      THCTensor *output_tmp = THCTensor_(newWithSize1d)(state, input->size(0));
      THCTensor_(resize0d)(state, output);

      cunn_MultiLabelMarginCriterion_updateOutput_kernel<scalar_t, accreal>
        <<<blocks, threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
          THCTensor_(data)(state, output_tmp),
          THCTensor_(data)(state, input),
          THCIndexTensor_(data)(state, target),
          THCTensor_(data)(state, istarget),
          nframe, dim,
          reduction == at::Reduction::Mean
          );
      THCudaCheck(cudaGetLastError());
      THCTensor_(set1d)(state, output, 0, ScalarConvert<accreal, scalar_t>::to(THCTensor_(sumall)(state, output_tmp)));
      THCTensor_(free)(state, output_tmp);
    }
    else
    {
      THCTensor_(resize1d)(state, output, input->size(0));

      cunn_MultiLabelMarginCriterion_updateOutput_kernel<scalar_t, accreal>
        <<<blocks, threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
          THCTensor_(data)(state, output),
          THCTensor_(data)(state, input),
          THCIndexTensor_(data)(state, target),
          THCTensor_(data)(state, istarget),
          nframe, dim,
          false
          );
      THCudaCheck(cudaGetLastError());
    }
  }
  else {
    TORCH_INTERNAL_ASSERT(false, "non-empty vector or matrix expected (shouldn't get here)");
  }

  THCTensor_(free)(state, input);
  THCIndexTensor_(free)(state, target);
  THCTensor_(free)(state, istarget);
  #endif // THC_REAL_IS_BFLOAT16 && !__HIP_PLATFORM_HCC__
}

void THNN_(MultiLabelMarginCriterion_updateGradInput)(
            THCState *state,
            THCTensor *input,
            THCIndexTensor *target,
            THCTensor *gradOutput,
            THCTensor *gradInput,
            THCTensor *istarget,
            int64_t reduction)
{
  #if defined(THC_REAL_IS_BFLOAT16) && !defined(__HIP_PLATFORM_HCC__)
  TORCH_CHECK(false, "MultiLabelMarginCriterion_updateGradInput not suppported with BFloat16");
  #else
  input = THCTensor_(newContiguous)(state, input);
  target = THCIndexTensor_(newContiguous)(state, target);
  istarget = THCTensor_(newContiguous)(state, istarget);
  gradOutput = THCTensor_(newContiguous)(state, gradOutput);
  THCTensor_(resizeAs)(state, gradInput, input);

  if(gradInput->dim() <= 1)
  {
    int dim = gradInput->dim() == 0 ? 1 : gradInput->size(0);
    int target_size = target->dim() == 0 ? 1 : target->size(0);
    THArgCheck(!target->is_empty() && (target->dim() <= 1) && (target_size == dim), 3,
               "inconsistent target size");
    TORCH_CHECK(target->sizes() == istarget->sizes(), "inconsistent isTarget size");
    dim3 blocks(1);
    dim3 threads(MULTILABELMARGIN_THREADS);

    cunn_MultiLabelMarginCriterion_updateGradInput_kernel<scalar_t, accreal>
      <<<blocks, threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, gradOutput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        THCTensor_(data)(state, istarget),
        1, dim,
        reduction == at::Reduction::Mean,
        reduction != at::Reduction::None);

  }
  else if(gradInput->dim() == 2)
  {
    int nframe = gradInput->size(0);
    int dim = gradInput->size(1);
    THArgCheck(!target->is_empty() && (target->dim() == 2) && (target->size(0) == nframe)
               && (target->size(1) == dim), 3, "inconsistent target size");
    THArgCheck(!istarget->is_empty() && (istarget->dim() == 2) && (istarget->size(0) == nframe)
               && (istarget->size(1) == dim), 3, "inconsistent isTarget size");
    dim3 blocks(gradInput->size(0));
    dim3 threads(MULTILABELMARGIN_THREADS);

    cunn_MultiLabelMarginCriterion_updateGradInput_kernel<scalar_t, accreal>
      <<<blocks, threads, 0, c10::cuda::getCurrentCUDAStream()>>>(
        THCTensor_(data)(state, gradInput),
        THCTensor_(data)(state, gradOutput),
        THCTensor_(data)(state, input),
        THCIndexTensor_(data)(state, target),
        THCTensor_(data)(state, istarget),
        gradInput->size(0), gradInput->size(1),
        reduction == at::Reduction::Mean,
        reduction != at::Reduction::None);
  }
  else {
    AT_ERROR("non-empty vector or matrix expected, got size: ", gradInput->sizes());
  }

  THCudaCheck(cudaGetLastError());

  THCTensor_(free)(state, input);
  THCIndexTensor_(free)(state, target);
  THCTensor_(free)(state, istarget);
  THCTensor_(free)(state, gradOutput);
  #endif // THC_REAL_IS_BFLOAT16 && !__HIP_PLATFORM_HCC__
}

#endif
