#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/native/quantized/fake_quant_affine.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>
#include <cmath>

/* Fake quantize a tensor
Args:
  output: output tensor.
  input : input tensor.
  sc:  scale to quantize the input tensor to
  zero_point: zero_point
  quant_min: minimum quantized value
  quant_max: maximum quantized value
Returns:
  Fake quantized tensor (float dtype).
*/
namespace at {
namespace native {
void fake_quantize_tensor_kernel_cuda(
    Tensor& output,
    const Tensor& input,
    float scale,
    int64_t zero_point,
    int64_t quant_min,
    int64_t quant_max) {
  // scalar type of this function is guaranteed to be float
  float inv_scale = 1.0f / scale;
  auto iter = TensorIteratorConfig()
    .check_all_same_dtype(false)
    .add_output(output)
    .add_input(input)
    .build();
  gpu_kernel(iter,
    [=] GPU_LAMBDA (float input_val) -> float {
      return (fminf(
                quant_max,
                fmaxf(
                    quant_min,
                    static_cast<int64_t>(std::nearbyint(
                        input_val * inv_scale + zero_point)))) -
            zero_point) *
          scale;
    });
}

void fake_quantize_grad_tensor_kernel_cuda(
    Tensor& input_grad,
    const Tensor& input,
    const Tensor& output_grad,
    float scale,
    int64_t zero_point,
    int64_t quant_min,
    int64_t quant_max) {
  // scalar type of this function is guaranteed to be float
  float inv_scale = 1.0f / scale;
  auto iter = TensorIteratorConfig()
    .check_all_same_dtype(false)
    .add_output(input_grad)
    .add_input(output_grad)
    .add_input(input)
    .build();
  gpu_kernel(iter,
    [=] GPU_LAMBDA (float dy, float x) -> float {
      int64_t Xq = std::nearbyint(x * inv_scale + zero_point);
      return (Xq >= quant_min && Xq <= quant_max) * dy;
    });
}

REGISTER_DISPATCH(fake_quant_tensor_stub, &fake_quantize_tensor_kernel_cuda);
REGISTER_DISPATCH(fake_quant_grad_tensor_stub, &fake_quantize_grad_tensor_kernel_cuda);

// Fake quantize per channel

void fake_quant_per_channel_cuda(TensorIterator &iter, int64_t quant_min, int64_t quant_max) {
  gpu_kernel(iter,
    [=] GPU_LAMBDA (float input_val, float scale, int64_t zero_point) -> float {
      float inv_scale = 1.0f / scale;
      return (fminf(
                quant_max,
                fmaxf(
                    quant_min,
                    static_cast<int64_t>(std::nearbyint(
                        input_val * inv_scale + zero_point)))) -
            zero_point) *
          scale;
    });
}

void fake_quant_grad_per_channel_cuda(TensorIterator &iter, int64_t quant_min, int64_t quant_max) {
  gpu_kernel(iter,
    [=] GPU_LAMBDA (float x, float dy, float scale, int64_t zero_point) -> float {
      float inv_scale = 1.0f / scale;
      int64_t Xq = std::nearbyint(x * inv_scale + zero_point);
      return (Xq >= quant_min && Xq <= quant_max) * dy;
    });
}

REGISTER_DISPATCH(fake_quant_per_channel_stub, &fake_quant_per_channel_cuda);
REGISTER_DISPATCH(fake_quant_grad_per_channel_stub, &fake_quant_grad_per_channel_cuda);

} // namespace native
} // namespace at
