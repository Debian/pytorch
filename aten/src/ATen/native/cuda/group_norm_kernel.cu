#include "hip/hip_runtime.h"
#include <ATen/native/group_norm.h>

#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/native/cuda/block_reduce.cuh>
#include <THC/THCDeviceUtils.cuh>

#include <c10/cuda/CUDAMathCompat.h>

namespace at {
namespace native {

namespace {

constexpr int kCUDANumThreads = 256;
constexpr int kReduceTileSize = 32;

template <typename T>
__global__ void RowwiseMomentsCUDAKernel(
    int64_t N,
    T eps,
    const T* X,
    T* mean,
    T* rstd) {
  using T_ACC = acc_type<T, true>;
  __shared__ T_ACC m_shared[C10_WARP_SIZE];
  __shared__ T_ACC v_shared[C10_WARP_SIZE];
  const int64_t i = blockIdx.x;
  T_ACC sum1 = 0;
  T_ACC sum2 = 0;
  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    const int64_t index = i * N + j;
    sum1 += static_cast<T_ACC>(X[index]);
    sum2 += static_cast<T_ACC>(X[index]) * static_cast<T_ACC>(X[index]);
  }
  sum1 = cuda_utils::BlockReduceSum<T_ACC>(sum1, m_shared);
  sum2 = cuda_utils::BlockReduceSum<T_ACC>(sum2, v_shared);
  if (threadIdx.x == 0) {
    const T_ACC scale = T_ACC(1) / static_cast<T_ACC>(N);
    sum1 *= scale;
    sum2 = c10::cuda::compat::max(sum2 * scale - sum1 * sum1, T_ACC(0));
    mean[i] = sum1;
    rstd[i] = c10::cuda::compat::rsqrt(sum2 + static_cast<T_ACC>(eps));
  }
}

template <typename T>
__global__ void ComputeFusedParamsCUDAKernel(
    int64_t N,
    int64_t C,
    int64_t group,
    const T* mean,
    const T* rstd,
    const T* gamma,
    const T* beta,
    acc_type<T, true>* a,
    acc_type<T, true>* b) {
  using T_ACC = acc_type<T, true>;
  const int64_t index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < N * C) {
    const int64_t ng = index / (C / group);
    const int64_t c = index % C;
    const T_ACC x = (gamma == nullptr)
        ? static_cast<T_ACC>(rstd[ng])
        : static_cast<T_ACC>(rstd[ng]) * static_cast<T_ACC>(gamma[c]);
    a[index] = x;
    b[index] = -x * static_cast<T_ACC>(mean[ng]) +
        (beta == nullptr ? T_ACC(0) : static_cast<T_ACC>(beta[c]));
  }
}

template <typename T>
__global__ void GroupNormForwardSimpleCUDAKernel(
    int64_t N,
    int64_t C,
    int64_t HxW,
    const T* X,
    const acc_type<T, true>* a,
    const acc_type<T, true>* b,
    T* Y) {
  using T_ACC = acc_type<T, true>;
  const int64_t index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < N * C * HxW) {
    const int64_t nc = index / HxW;
    Y[index] = a[nc] * static_cast<T_ACC>(X[index]) + b[nc];
  }
}

template <typename T>
__global__ void GroupNormForwardCUDAKernel(
    int64_t HxW,
    const T* X,
    const acc_type<T, true>* a,
    const acc_type<T, true>* b,
    T* Y) {
  using T_ACC = acc_type<T, true>;
  const int64_t nc = blockIdx.x;
  for (int64_t hw = threadIdx.x; hw < HxW; hw += blockDim.x) {
    const int64_t index = nc * HxW + hw;
    Y[index] = a[nc] * static_cast<T_ACC>(X[index]) + b[nc];
  }
}

template <typename T>
__global__ void ComputeInternalGradientsCUDAKernel(
    int64_t HxW,
    const T* dY,
    const T* X,
    acc_type<T, true>* ds,
    acc_type<T, true>* db) {
  using T_ACC = acc_type<T, true>;
  __shared__ T_ACC ds_shared[C10_WARP_SIZE];
  __shared__ T_ACC db_shared[C10_WARP_SIZE];
  const int64_t nc = blockIdx.x;
  T_ACC sum1 = 0;
  T_ACC sum2 = 0;
  for (int64_t hw = threadIdx.x; hw < HxW; hw += blockDim.x) {
    const int64_t index = nc * HxW + hw;
    sum1 += static_cast<T_ACC>(dY[index]) * static_cast<T_ACC>(X[index]);
    sum2 += static_cast<T_ACC>(dY[index]);
  }
  sum1 = cuda_utils::BlockReduceSum<T_ACC>(sum1, ds_shared);
  sum2 = cuda_utils::BlockReduceSum<T_ACC>(sum2, db_shared);
  if (threadIdx.x == 0) {
    ds[nc] = sum1;
    db[nc] = sum2;
  }
}

template <typename T>
__global__ void ComputeGradOutputCoeffientCUDAKernel(
    int64_t N,
    int64_t C,
    int64_t group,
    const T* rstd,
    const T* gamma,
    acc_type<T, true>* c1) {
  using T_ACC = acc_type<T, true>;
  const int64_t nc = blockIdx.x * blockDim.x + threadIdx.x;
  if (nc < N * C) {
    const int64_t ng = nc / (C / group);
    const int64_t c = nc % C;
    c1[nc] = static_cast<T_ACC>(rstd[ng]) *
        (gamma == nullptr ? T_ACC(1) : static_cast<T_ACC>(gamma[c]));
  }
}

template <typename T>
__global__ void ComputeBackwardFusedParamsCUDAKernel(
    int64_t C,
    int64_t HxW,
    int64_t group,
    const T* mean,
    const T* rstd,
    const T* gamma,
    const acc_type<T, true>* ds,
    const acc_type<T, true>* db,
    acc_type<T, true>* c2,
    acc_type<T, true>* c3) {
  using T_ACC = acc_type<T, true>;
  __shared__ T_ACC ds_shared[C10_WARP_SIZE];
  __shared__ T_ACC db_shared[C10_WARP_SIZE];
  const int64_t G = group;
  const int64_t D = C / G;
  const int64_t n = blockIdx.x;
  const int64_t g = blockIdx.y;
  const int64_t ng = n * G + g;
  T_ACC sum1 = 0;
  T_ACC sum2 = 0;
  for (int64_t i = threadIdx.x; i < D; i += blockDim.x) {
    const int64_t index = ng * D + i;
    const int64_t c = g * D + i;
    const T_ACC gamma_v =
        gamma == nullptr ? T_ACC(1) : static_cast<T_ACC>(gamma[c]);
    sum1 += ds[index] * gamma_v;
    sum2 += db[index] * gamma_v;
  }
  sum1 = cuda_utils::BlockReduceSum<T_ACC>(sum1, ds_shared);
  sum2 = cuda_utils::BlockReduceSum<T_ACC>(sum2, db_shared);
  if (threadIdx.x == 0) {
    const T_ACC s = T_ACC(1) / static_cast<T_ACC>(D * HxW);
    const T_ACC x = (sum2 * static_cast<T_ACC>(mean[ng]) - sum1) *
        static_cast<T_ACC>(rstd[ng]) * static_cast<T_ACC>(rstd[ng]) *
        static_cast<T_ACC>(rstd[ng]) * s;
    c2[ng] = x;
    c3[ng] = -x * static_cast<T_ACC>(mean[ng]) -
        sum2 * static_cast<T_ACC>(rstd[ng]) * s;
  }
}

template <typename T>
__global__ void GroupNormBackwardSimpleCUDAKernel(
    int64_t N,
    int64_t C,
    int64_t HxW,
    int64_t group,
    const T* dY,
    const T* X,
    const acc_type<T, true>* c1,
    const acc_type<T, true>* c2,
    const acc_type<T, true>* c3,
    T* dX) {
  using T_ACC = acc_type<T, true>;
  const int64_t index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < N * C * HxW) {
    const int64_t nc = index / HxW;
    const int64_t ng = nc / (C / group);
    dX[index] = c1[nc] * static_cast<T_ACC>(dY[index]) +
        c2[ng] * static_cast<T_ACC>(X[index]) + c3[ng];
  }
}

template <typename T>
__global__ void GroupNormBackwardCUDAKernel(
    int64_t C,
    int64_t HxW,
    int64_t group,
    const T* dY,
    const T* X,
    const acc_type<T, true>* c1,
    const acc_type<T, true>* c2,
    const acc_type<T, true>* c3,
    T* dX) {
  using T_ACC = acc_type<T, true>;
  const int64_t D = C / group;
  const int64_t nc = blockIdx.x;
  const int64_t ng = nc / D;
  for (int64_t hw = threadIdx.x; hw < HxW; hw += blockDim.x) {
    const int64_t index = nc * HxW + hw;
    dX[index] = c1[nc] * static_cast<T_ACC>(dY[index]) +
        c2[ng] * static_cast<T_ACC>(X[index]) + c3[ng];
  }
}

template <typename T>
__global__ void GammaBetaBackwardSimpleCUDAKernel(
    int64_t N,
    int64_t C,
    int64_t group,
    const T* mean,
    const T* rstd,
    const acc_type<T, true>* ds,
    const acc_type<T, true>* db,
    T* dgamma,
    T* dbeta) {
  using T_ACC = acc_type<T, true>;
  const int64_t c = blockIdx.x * blockDim.x + threadIdx.x;
  if (c < C) {
    const int64_t G = group;
    const int64_t D = C / G;
    T_ACC sum1 = 0;
    T_ACC sum2 = 0;
    for (int64_t n = 0; n < N; ++n) {
      const int64_t nc = n * C + c;
      const int64_t ng = n * G + c / D;
      sum1 += (dgamma == nullptr)
          ? T_ACC(0)
          : ((ds[nc] - db[nc] * static_cast<T_ACC>(mean[ng])) *
             static_cast<T_ACC>(rstd[ng]));
      sum2 += (dbeta == nullptr) ? T_ACC(0) : db[nc];
    }
    if (dgamma != nullptr) {
      dgamma[c] = sum1;
    }
    if (dbeta != nullptr) {
      dbeta[c] = sum2;
    }
  }
}

template <typename T>
__global__ void GammaBetaBackwardCUDAKernel(
    int64_t N,
    int64_t C,
    int64_t group,
    const T* mean,
    const T* rstd,
    const acc_type<T, true>* ds,
    const acc_type<T, true>* db,
    T* dgamma,
    T* dbeta) {
  using T_ACC = acc_type<T, true>;
  __shared__ T_ACC g_shared[kReduceTileSize][kReduceTileSize + 1];
  __shared__ T_ACC b_shared[kReduceTileSize][kReduceTileSize + 1];
  const int64_t c = blockIdx.x * blockDim.x + threadIdx.x;
  T_ACC dg_sum1 = 0;
  T_ACC dg_sum2 = 0;
  T_ACC db_sum1 = 0;
  T_ACC db_sum2 = 0;
  if (c < C) {
    const int64_t G = group;
    const int64_t D = C / G;
    for (int64_t n = threadIdx.y; n < N; n += blockDim.y * 2) {
      const int64_t n1 = n;
      const int64_t n2 = n + blockDim.y;
      const int64_t nc1 = n1 * C + c;
      const int64_t nc2 = n2 * C + c;
      const int64_t ng1 = n1 * G + c / D;
      const int64_t ng2 = n2 * G + c / D;
      dg_sum1 += dgamma == nullptr
          ? T_ACC(0)
          : ((ds[nc1] - db[nc1] * static_cast<T_ACC>(mean[ng1])) *
             static_cast<T_ACC>(rstd[ng1]));
      db_sum1 += dbeta == nullptr ? T_ACC(0) : db[nc1];
      if (n2 < N) {
        dg_sum2 += dgamma == nullptr
            ? T_ACC(0)
            : ((ds[nc2] - db[nc2] * static_cast<T_ACC>(mean[ng2])) *
               static_cast<T_ACC>(rstd[ng2]));
        db_sum2 += dbeta == nullptr ? T_ACC(0) : db[nc2];
      }
    }
  }
  g_shared[threadIdx.y][threadIdx.x] = dg_sum1;
  g_shared[threadIdx.y + blockDim.y][threadIdx.x] = dg_sum2;
  b_shared[threadIdx.y][threadIdx.x] = db_sum1;
  b_shared[threadIdx.y + blockDim.y][threadIdx.x] = db_sum2;
  __syncthreads();
  T_ACC sum1 = g_shared[threadIdx.x][threadIdx.y];
  T_ACC sum2 = b_shared[threadIdx.x][threadIdx.y];
  sum1 = cuda_utils::WarpReduceSum<T_ACC>(sum1);
  sum2 = cuda_utils::WarpReduceSum<T_ACC>(sum2);
  if (threadIdx.x == 0) {
    const int64_t c = blockIdx.x * blockDim.x + threadIdx.y;
    if (c < C) {
      if (dgamma != nullptr) {
        dgamma[c] = sum1;
      }
      if (dbeta != nullptr) {
        dbeta[c] = sum2;
      }
    }
  }
  sum1 = g_shared[threadIdx.x][threadIdx.y + blockDim.y];
  sum2 = b_shared[threadIdx.x][threadIdx.y + blockDim.y];
  sum1 = cuda_utils::WarpReduceSum<T_ACC>(sum1);
  sum2 = cuda_utils::WarpReduceSum<T_ACC>(sum2);
  if (threadIdx.x == 0) {
    const int64_t c = blockIdx.x * blockDim.x + threadIdx.y + blockDim.y;
    if (c < C) {
      if (dgamma != nullptr) {
        dgamma[c] = sum1;
      }
      if (dbeta != nullptr) {
        dbeta[c] = sum2;
      }
    }
  }
}

template <typename T>
void GroupNormKernelImplInternal(
    const Tensor& X,
    const Tensor& gamma,
    const Tensor& beta,
    int64_t N,
    int64_t C,
    int64_t HxW,
    int64_t group,
    T eps,
    Tensor* Y,
    Tensor* mean,
    Tensor* rstd) {
  using T_ACC = acc_type<T, true>;
  TORCH_CHECK(X.numel() == N * C * HxW);
  TORCH_CHECK(!gamma.defined() || gamma.numel() == C);
  TORCH_CHECK(!beta.defined() || beta.numel() == C);
  if (N == 0) {
    return;
  }
  const int64_t G = group;
  const int64_t D = C / G;
  const T* X_data = X.data_ptr<T>();
  const T* gamma_data = gamma.defined() ? gamma.data_ptr<T>() : nullptr;
  const T* beta_data = beta.defined() ? beta.data_ptr<T>() : nullptr;
  T* Y_data = Y->data_ptr<T>();
  T* mean_data = mean->data_ptr<T>();
  T* rstd_data = rstd->data_ptr<T>();
  const auto kAccType = X.scalar_type() == kHalf ? kFloat : X.scalar_type();
  Tensor a = at::empty({N, C}, X.options().dtype(kAccType));
  Tensor b = at::empty({N, C}, X.options().dtype(kAccType));
  T_ACC* a_data = a.data_ptr<T_ACC>();
  T_ACC* b_data = b.data_ptr<T_ACC>();
  hipStream_t hip_stream = at::cuda::getCurrentCUDAStream();
  RowwiseMomentsCUDAKernel<T>
      <<<N * G, cuda_utils::kCUDABlockReduceNumThreads, 0, hip_stream>>>(
          D * HxW, eps, X_data, mean_data, rstd_data);
  int64_t B = (N * C + kCUDANumThreads - 1) / kCUDANumThreads;
  ComputeFusedParamsCUDAKernel<T><<<B, kCUDANumThreads, 0, hip_stream>>>(
      N, C, G, mean_data, rstd_data, gamma_data, beta_data, a_data, b_data);
  if (HxW < kCUDANumThreads) {
    B = (N * C * HxW + kCUDANumThreads - 1) / kCUDANumThreads;
    GroupNormForwardSimpleCUDAKernel<T><<<B, kCUDANumThreads, 0, hip_stream>>>(
        N, C, HxW, X_data, a_data, b_data, Y_data);
  } else {
    GroupNormForwardCUDAKernel<T><<<N * C, kCUDANumThreads, 0, hip_stream>>>(
        HxW, X_data, a_data, b_data, Y_data);
  }
  AT_CUDA_CHECK(hipGetLastError());
}

void GroupNormKernelImpl(
    const Tensor& X,
    const Tensor& gamma,
    const Tensor& beta,
    int64_t N,
    int64_t C,
    int64_t HxW,
    int64_t group,
    double eps,
    Tensor* Y,
    Tensor* mean,
    Tensor* rstd) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      X.scalar_type(),
      "GroupNormKernelImpl",
      [&]() {
        AT_SKIP_BFLOAT16_IF_NOT_ROCM(scalar_t, "GroupNormKernelImpl", [&]() {
          GroupNormKernelImplInternal<scalar_t>(
              X,
              gamma,
              beta,
              N,
              C,
              HxW,
              group,
              static_cast<scalar_t>(eps),
              Y,
              mean,
              rstd);
        });
      });
}

template <typename T>
void GroupNormBackwardKernelImplInternal(
    const Tensor& dY,
    const Tensor& X,
    const Tensor& mean,
    const Tensor& rstd,
    const Tensor& gamma,
    int64_t N,
    int64_t C,
    int64_t HxW,
    int64_t group,
    Tensor* dX,
    Tensor* dgamma,
    Tensor* dbeta) {
  using T_ACC = acc_type<T, true>;
  const int64_t G = group;
  TORCH_CHECK(dY.numel() == N * C * HxW);
  TORCH_CHECK(X.numel() == N * C * HxW);
  TORCH_CHECK(mean.numel() == N * G);
  TORCH_CHECK(rstd.numel() == N * G);
  TORCH_CHECK(!gamma.defined() || gamma.numel() == C);
  hipStream_t hip_stream = at::cuda::getCurrentCUDAStream();

  if (N == 0) {
    if (dgamma->defined()) {
      T* dgamma_data = dgamma->data_ptr<T>();
      AT_CUDA_CHECK(hipMemsetAsync(
          dgamma_data, 0, dgamma->numel() * sizeof(T), hip_stream));
    }
    if (dbeta->defined()) {
      T* dbeta_data = dbeta->data_ptr<T>();
      AT_CUDA_CHECK(hipMemsetAsync(
          dbeta_data, 0, dbeta->numel() * sizeof(T), hip_stream));
    }
    return;
  }

  const T* dY_data = dY.data_ptr<T>();
  const T* X_data = X.data_ptr<T>();
  const T* mean_data = mean.data_ptr<T>();
  const T* rstd_data = rstd.data_ptr<T>();
  const T* gamma_data = gamma.defined() ? gamma.data_ptr<T>() : nullptr;
  T* dX_data = dX->defined() ? dX->data_ptr<T>() : nullptr;
  const auto kAccType = X.scalar_type() == kHalf ? kFloat : X.scalar_type();
  Tensor ds = at::empty({N, C}, X.options().dtype(kAccType));
  Tensor db = at::empty({N, C}, X.options().dtype(kAccType));
  T_ACC* ds_data = ds.data_ptr<T_ACC>();
  T_ACC* db_data = db.data_ptr<T_ACC>();
  ComputeInternalGradientsCUDAKernel<T>
      <<<N * C, cuda_utils::kCUDABlockReduceNumThreads, 0, hip_stream>>>(
          HxW, dY_data, X_data, ds_data, db_data);
  if (dX != nullptr) {
    Tensor c1 = at::empty({N, C}, X.options().dtype(kAccType));
    Tensor c2 = at::empty({N, G}, X.options().dtype(kAccType));
    Tensor c3 = at::empty({N, G}, X.options().dtype(kAccType));
    T_ACC* c1_data = c1.data_ptr<T_ACC>();
    T_ACC* c2_data = c2.data_ptr<T_ACC>();
    T_ACC* c3_data = c3.data_ptr<T_ACC>();
    int64_t B = (N * C + kCUDANumThreads - 1) / kCUDANumThreads;
    ComputeGradOutputCoeffientCUDAKernel<T>
        <<<B, kCUDANumThreads, 0, hip_stream>>>(
            N, C, G, rstd_data, gamma_data, c1_data);
    ComputeBackwardFusedParamsCUDAKernel<T>
        <<<dim3(N, G),
           cuda_utils::kCUDABlockReduceNumThreads,
           0,
           hip_stream>>>(
            C,
            HxW,
            G,
            mean_data,
            rstd_data,
            gamma_data,
            ds_data,
            db_data,
            c2_data,
            c3_data);
    if (HxW < kCUDANumThreads) {
      B = (N * C * HxW + kCUDANumThreads - 1) / kCUDANumThreads;
      GroupNormBackwardSimpleCUDAKernel<
          T><<<B, kCUDANumThreads, 0, hip_stream>>>(
          N, C, HxW, G, dY_data, X_data, c1_data, c2_data, c3_data, dX_data);
    } else {
      GroupNormBackwardCUDAKernel<T>
          <<<N * C, kCUDANumThreads, 0, hip_stream>>>(
              C, HxW, G, dY_data, X_data, c1_data, c2_data, c3_data, dX_data);
    }
  }
  if (dgamma->defined() || dbeta->defined()) {
    T* dgamma_data = dgamma->defined() ? dgamma->data_ptr<T>() : nullptr;
    T* dbeta_data = dbeta->defined() ? dbeta->data_ptr<T>() : nullptr;
    if (N < 512) {
      // For small batch size, do colwise reduce directly.
      const int64_t B = (C + kCUDANumThreads - 1) / kCUDANumThreads;
      GammaBetaBackwardSimpleCUDAKernel<T>
          <<<B, kCUDANumThreads, 0, hip_stream>>>(
              N,
              C,
              G,
              mean_data,
              rstd_data,
              ds_data,
              db_data,
              dgamma_data,
              dbeta_data);
    } else {
      const int64_t B = (C + kReduceTileSize - 1) / kReduceTileSize;
      constexpr int kThreadX = kReduceTileSize;
      constexpr int kThreadY = kReduceTileSize / 2;
      GammaBetaBackwardCUDAKernel<T>
          <<<B, dim3(kThreadX, kThreadY), 0, hip_stream>>>(
              N,
              C,
              G,
              mean_data,
              rstd_data,
              ds_data,
              db_data,
              dgamma_data,
              dbeta_data);
    }
  }
  AT_CUDA_CHECK(hipGetLastError());
}

void GroupNormBackwardKernelImpl(
    const Tensor& dY,
    const Tensor& X,
    const Tensor& mean,
    const Tensor& rstd,
    const Tensor& gamma,
    int64_t N,
    int64_t C,
    int64_t HxW,
    int64_t group,
    Tensor* dX,
    Tensor* dgamma,
    Tensor* dbeta) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      X.scalar_type(),
      "GroupNormBackwardKernelImpl",
      [&]() {
        AT_SKIP_BFLOAT16_IF_NOT_ROCM(
            scalar_t, "GroupNormBackwardKernelImpl", [&]() {
              GroupNormBackwardKernelImplInternal<scalar_t>(
                  dY,
                  X,
                  mean,
                  rstd,
                  gamma,
                  N,
                  C,
                  HxW,
                  group,
                  dX,
                  dgamma,
                  dbeta);
            });
      });
}

} // namespace

REGISTER_DISPATCH(GroupNormKernel, &GroupNormKernelImpl);
REGISTER_DISPATCH(GroupNormBackwardKernel, &GroupNormBackwardKernelImpl);

} // namespace native
} // namespace at
