#include "hip/hip_runtime.h"
#include <ATen/Dispatch.h>
#include <ATen/ExpandUtils.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/CUDAGenerator.h>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/DistributionTemplates.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>
#include <utility>
#include <functional>

#include <ATen/native/Distributions.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/LegacyTHFunctionsCUDA.h>

#include <THC/THCGeneral.h>
#include <THC/THCApply.cuh>
#include <THC/THCDeviceUtils.cuh>

#include <cstdint>
#include <limits>
#include <utility>
#include <type_traits>

namespace at { namespace native {

void geometric_kernel_cuda(TensorIterator& iter, double p_, Generator* gen_) {
  auto gen = get_generator_or_default<CUDAGenerator>(gen_, cuda::detail::getDefaultCUDAGenerator());
  AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "geometric_cuda", [&] {
    if (std::is_same<scalar_t, double>::value) {
      // define lambda for geometric transformation
      auto geometric_func = [p_] __device__ (double rand) {
        return static_cast<scalar_t>(::ceil(::log(rand) / ::log(static_cast<double>(1.0)-p_)));
      };
      distribution_nullary_kernel<scalar_t, double, curand4_engine_calls/2>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_uniform2_double(state); },
        geometric_func);
    } else {
      auto p = static_cast<float>(p_);
      auto geometric_func = [p] __device__ (float rand) {
        // use __logf fast approximation for peak bandwidth
        return static_cast<scalar_t>(::ceil(__logf(rand) / __logf(static_cast<float>(1.0)-p)));
      };
      distribution_nullary_kernel<scalar_t, float, curand4_engine_calls>(iter,
        gen,
        [] __device__ (hiprandStatePhilox4_32_10_t* state) { return hiprand_uniform4(state); },
        geometric_func);
    }
   });
}

REGISTER_DISPATCH(geometric_stub, &geometric_kernel_cuda);

}} // namespace at::native
