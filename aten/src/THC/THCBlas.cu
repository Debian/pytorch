#include "hip/hip_runtime.h"
#include <THC/THCBlas.h>
#include <THC/THCGeneral.h>
#include <TH/THHalf.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDABlas.h>

#include <algorithm>
#include <mutex>

#ifdef __HIP_PLATFORM_HCC__
#include <hip/hip_version.h>
#endif

float THCudaBlas_Sdot(THCState *state, int64_t n, float *x, int64_t incx, float *y, int64_t incy)
{
  if (n == 1) {
    incx = 1;
    incy = 1;
  }

  if ((n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX)) {
    int i_n = (int)n;
    int i_incx = (int)incx;
    int i_incy = (int)incy;
    float result;
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
    THCublasCheck(hipblasSdot(handle, i_n, x, i_incx, y, i_incy, &result));
    return result;
  }

  THError("Cublas_Sdot only supports n, incx and incy "
          "up to signed integer limits: %d", INT_MAX);
  return 0;
}

double THCudaBlas_Ddot(THCState *state, int64_t n, double *x, int64_t incx, double *y, int64_t incy)
{
  if (n == 1) {
    incx = 1;
    incy = 1;
  }

  if ((n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX)) {
    int i_n = (int)n;
    int i_incx = (int)incx;
    int i_incy = (int)incy;
    double result;
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
    THCublasCheck(hipblasDdot(handle, i_n, x, i_incx, y, i_incy, &result));
    return result;
  }

  THError("Cublas_Ddot only supports n, incx and incy "
          "up to signed integer limits: %d", INT_MAX);
  return 0;
}

at::Half THCudaBlas_Hdot(THCState *state, int64_t n, at::Half *x, int64_t incx, at::Half *y, int64_t incy)
{
#if CUDA_VERSION >= 8000
  if (n == 1) {
    incx = 1;
    incy = 1;
  }

  if ((n <= INT_MAX) && (incx <= INT_MAX) && (incy <= INT_MAX)) {
    at::Half result;
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
    THCublasCheck(hipblasDotEx(handle, n,
                              x, HIP_R_16F, incx,
                              y, HIP_R_16F, incy,
                              &result, HIP_R_16F,
                              HIP_R_32F));
    return result;
  }

  THError("Cublas_Hdot only supports n, incx and incy "
          "up to signed integer limits: %d", INT_MAX);
  return 0.0;
#elif HIP_VERSION >= 210
  if (n == 1) {
    incx = 1;
    incy = 1;
  }

  at::Half result;
  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
  hipblasSetStream(handle, at::cuda::getCurrentCUDAStream().stream());
  THCublasCheck(rocblas_hdot(handle, n,
                             reinterpret_cast<rocblas_half*>(x), incx,
                             reinterpret_cast<rocblas_half*>(y), incy,
                             reinterpret_cast<rocblas_half*>(&result)));
  return result;
#else
  THError("Cublas_Hdot requires CUDA 8.0+");
  return 0.0;
#endif
}

/* Level 2 */

void adjustLdLevel2(int64_t m, int64_t n, int64_t *lda)
{
  // Note: leading dimensions generally are checked that they are > 0 and at least as big the result
  // requires (even if the value won't be used).
  // TODO: why does Level3 check trans but this doesn't?
  if (n <= 1)
    *lda = std::max<int64_t>(m, 1);
}

void THCudaBlas_Sger(THCState *state, int64_t m, int64_t n, float alpha, float *x, int64_t incx, float *y, int64_t incy, float *a, int64_t lda)
{
  adjustLdLevel2(m, n, &lda);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (lda <= INT_MAX)  && (incx <= INT_MAX) && (incy <= INT_MAX) )
    {
      int i_m = (int)m;
      int i_n = (int)n;
      int i_lda = (int)lda;
      int i_incx = (int)incx;
      int i_incy = (int)incy;

      hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
      THCublasCheck(hipblasSger(handle, i_m, i_n, &alpha, x, i_incx, y, i_incy, a, i_lda));
      return;
    }
  THError("Cublas_Sger only supports m, n, lda, incx, incy"
          "with the bound [val] <= %d", INT_MAX);
}

void THCudaBlas_Dger(THCState *state, int64_t m, int64_t n, double alpha, double *x, int64_t incx, double *y, int64_t incy, double *a, int64_t lda)
{
  adjustLdLevel2(m, n, &lda);

  if( (m <= INT_MAX) && (n <= INT_MAX) && (lda <= INT_MAX)  && (incx <= INT_MAX) && (incy <= INT_MAX) )
    {
      int i_m = (int)m;
      int i_n = (int)n;
      int i_lda = (int)lda;
      int i_incx = (int)incx;
      int i_incy = (int)incy;

      hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
      THCublasCheck(hipblasDger(handle, i_m, i_n, &alpha, x, i_incx, y, i_incy, a, i_lda));
      return;
    }
  THError("Cublas_Dger only supports m, n, lda, incx, incy"
          "with the bound [val] <= %d", INT_MAX);
}


hipblasOperation_t convertTransToCublasOperation(char trans) {
  if (trans == 't') return HIPBLAS_OP_T;
  else if (trans == 'n') return HIPBLAS_OP_N;
  else if (trans == 'c') return HIPBLAS_OP_C;
  else {
    THError("trans must be one of: t, n, c");
    return HIPBLAS_OP_T;
  }
}

void adjustLdLevel3(char transa, char transb, int64_t m, int64_t n, int64_t k, int64_t *lda, int64_t *ldb, int64_t *ldc)
{
  int transa_ = ((transa == 't') || (transa == 'T'));
  int transb_ = ((transb == 't') || (transb == 'T'));

  // Note: leading dimensions generally are checked that they are > 0 and at least as big the result
  // requires (even if the value won't be used).
  if(n <= 1)
    *ldc = std::max<int64_t>(m, 1);

  if(transa_)
  {
    if(m <= 1)
      *lda = std::max<int64_t>(k, 1);
  }
  else
  {
    if(k <= 1)
      *lda = std::max<int64_t>(m, 1);
  }

  if(transb_)
  {
    if(k <= 1)
      *ldb = std::max<int64_t>(n, 1);
  }
  else
  {
    if(n <= 1)
      *ldb = std::max<int64_t>(k, 1);
  }

}

// Check https://github.com/pytorch/pytorch/issues/22078
// for information about the bug. We don't know the exact conditions that trigger it,
// but using Sgemm or Hgemm on Maxwell or Pascal seems to be a
// necessary condition.
static void checkCuda90Bug(int i_m, int i_n, int i_k)
{
#if CUDA_VERSION < 9200 && CUDA_VERSION >= 9000
  static std::once_flag alreadyWarned;
  const int LIMIT = 1 << 21;
  if (i_m > LIMIT || i_n > LIMIT || i_k > LIMIT) {
    hipDeviceProp_t* prop = at::cuda::getCurrentDeviceProperties();
    if (prop->major == 5 || prop->major == 6) {
      std::call_once(alreadyWarned, []() {
        TORCH_WARN("Matrix multiplication for dimensions larger than 2^21 has known bugs on your combination of CUDA version and device type. Please consider upgrading to CUDA 9.2 or later.");
      });
    }
  }
#endif
}

/* Level 3 */
void THCudaBlas_Sgemm(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k, float alpha, float *a, int64_t lda, float *b, int64_t ldb, float beta, float *c, int64_t ldc)
{
  checkCuda90Bug((int)m, (int)n, (int)k);
  at::cuda::blas::gemm<float>(transa, transb, m, n, k, alpha, a, lda, b, ldb, beta, c, ldc);
}

// In CUDA 8.0, definition of data types for sgemmex changed
#if CUDA_VERSION < 8000
#  define HIP_R_16F CUBLAS_DATA_HALF
#endif

void THCudaBlas_Hgemm(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k, at::Half alpha, at::Half *a, int64_t lda, at::Half *b, int64_t ldb, at::Half beta, at::Half *c, int64_t ldc)
{
  checkCuda90Bug((int)m, (int)n, (int)k);
  at::cuda::blas::gemm<at::Half>(transa, transb, m, n, k, alpha, a, lda, b, ldb, beta, c, ldc);
}

#ifdef __HIP_PLATFORM_HCC__
void THCudaBlas_Bgemm(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k, at::BFloat16 alpha, at::BFloat16 *a, int64_t lda, at::BFloat16 *b, int64_t ldb, at::BFloat16 beta, at::BFloat16 *c, int64_t ldc)
{
  at::cuda::blas::gemm<at::BFloat16>(transa, transb, m, n, k, alpha, a, lda, b, ldb, beta, c, ldc);
}
#endif

void THCudaBlas_Dgemm(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k, double alpha, double *a, int64_t lda, double *b, int64_t ldb, double beta, double *c, int64_t ldc)
{
  at::cuda::blas::gemm<double>(transa, transb, m, n, k, alpha, a, lda, b, ldb, beta, c, ldc);
}

#if CUDA_VERSION >= 9010  || defined __HIP_PLATFORM_HCC__
void THCudaBlas_HgemmStridedBatched(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k,
                             at::Half alpha, const at::Half *a, int64_t lda, int64_t strideA, const at::Half *b, int64_t ldb, int64_t strideB,
                             at::Half beta, at::Half *c, int64_t ldc, int64_t strideC, int64_t batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )

  {
    THError("Cublas_SgemmStridedBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLdLevel3(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
  float fAlpha = alpha;
  float fBeta = beta;
#ifdef __HIP_PLATFORM_HCC__
  THCublasCheck(rocblas_gemm_strided_batched_ex(handle, opa, opb, (int)m, (int)n, (int)k,
                                   (void*)&fAlpha, a, rocblas_datatype_f16_r, (int)lda, strideA,
                                   b, rocblas_datatype_f16_r, (int)ldb, strideB,
                                   (void*)&fBeta, c, rocblas_datatype_f16_r, (int)ldc, strideC,
                                   c, rocblas_datatype_f16_r, (int)ldc, strideC,
                                   (int) batchCount, rocblas_datatype_f32_r, rocblas_gemm_algo_standard,
                                   0, 0));
#else
  THCublasCheck(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));
  THCublasCheck(hipblasGemmStridedBatchedEx(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   (void*)&fAlpha, a, HIP_R_16F, (int)lda, strideA,
                                   b, HIP_R_16F, (int)ldb, strideB,
                                   (void*)&fBeta, c, HIP_R_16F, (int)ldc, strideC,
                                   (int)batchCount, HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
  THCublasCheck(hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH));
#endif // __HIP_PLATFORM_HCC__
}
#endif // CUDA_VERSION or __HIP_PLATFORM_HCC__

#ifdef __HIP_PLATFORM_HCC__
void THCudaBlas_BgemmStridedBatched(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k,
                             at::BFloat16 alpha, const at::BFloat16 *a, int64_t lda, int64_t strideA, const at::BFloat16 *b, int64_t ldb, int64_t strideB,
                             at::BFloat16 beta, at::BFloat16 *c, int64_t ldc, int64_t strideC, int64_t batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )

  {
    THError("Cublas_SgemmStridedBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLdLevel3(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
  float fAlpha = alpha;
  float fBeta = beta;
  THCublasCheck(rocblas_gemm_strided_batched_ex(handle, opa, opb, (int)m, (int)n, (int)k,
                                   (void*)&fAlpha, a, rocblas_datatype_bf16_r, (int)lda, strideA,
                                   b, rocblas_datatype_bf16_r, (int)ldb, strideB,
                                   (void*)&fBeta, c, rocblas_datatype_bf16_r, (int)ldc, strideC,
                                   c, rocblas_datatype_bf16_r, (int)ldc, strideC,
                                   (int) batchCount, rocblas_datatype_f32_r, rocblas_gemm_algo_standard,
                                   0, 0, NULL, NULL));
}
#endif // __HIP_PLATFORM_HCC__

void THCudaBlas_SgemmBatched(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k,
                             float alpha, const float *a[], int64_t lda, const float *b[], int64_t ldb,
                             float beta, float *c[], int64_t ldc, int64_t batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )
  {
    THError("Cublas_SgemmBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

#ifdef __HIP_PLATFORM_HCC__

  const int64_t stridea = (transa == 'N' || transa == 'n') ? lda*k : lda*n;
  const int64_t strideb = (transb == 'N' || transb == 'n') ? ldb*n : ldb*k;
  const int64_t stridec = ldc*n;

  THCudaBlas_SgemmStridedBatched(state, transa, transb, m, n, k, alpha, *a, lda, stridea, *b, ldb, strideb, beta, *c, ldc, stridec, batchCount);

#else

  adjustLdLevel3(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
  THCublasCheck(hipblasSgemmBatched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   &alpha, a, (int)lda, b, (int)ldb, &beta, c, (int)ldc,
                                   (int)batchCount));
#endif
}

#if CUDA_VERSION >= 8000 || defined __HIP_PLATFORM_HCC__
void THCudaBlas_SgemmStridedBatched(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k,
                             float alpha, const float *a, int64_t lda, int64_t strideA, const float *b, int64_t ldb, int64_t strideB,
                             float beta, float *c, int64_t ldc, int64_t strideC, int64_t batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )

  {
    THError("Cublas_SgemmStridedBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLdLevel3(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
  THCublasCheck(hipblasSgemmStridedBatched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   &alpha, a, (int)lda, strideA, b, (int)ldb, strideB, &beta, c, (int)ldc, strideC,
                                   (int)batchCount));
}
#endif

void THCudaBlas_DgemmBatched(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k,
                             double alpha, const double *a[], int64_t lda, const double *b[], int64_t ldb,
                             double beta, double *c[], int64_t ldc, int64_t batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )
  {
    THError("Cublas_DgemmBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

#ifdef __HIP_PLATFORM_HCC__

  const int64_t stridea = (transa == 'N' || transa == 'n') ? lda*k : lda*n;
  const int64_t strideb = (transb == 'N' || transb == 'n') ? ldb*n : ldb*k;
  const int64_t stridec = ldc*n;

  THCudaBlas_DgemmStridedBatched(state, transa, transb, m, n, k, alpha, *a, lda, stridea, *b, ldb, strideb, beta, *c, ldc, stridec, batchCount);

#else

  adjustLdLevel3(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
  THCublasCheck(hipblasDgemmBatched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   &alpha, a, (int)lda, b, (int)ldb, &beta, c, (int)ldc,
                                   (int)batchCount));
#endif
}

#if CUDA_VERSION >= 8000 || defined __HIP_PLATFORM_HCC__
void THCudaBlas_DgemmStridedBatched(THCState *state, char transa, char transb, int64_t m, int64_t n, int64_t k,
                             double alpha, const double *a, int64_t lda, int64_t strideA, const double *b, int64_t ldb, int64_t strideB,
                             double beta, double *c, int64_t ldc, int64_t strideC, int64_t batchCount)
{
  if( (m >= INT_MAX) || (n >= INT_MAX) || (k >= INT_MAX) || (lda >= INT_MAX)  || (ldb >= INT_MAX) || (ldc >= INT_MAX) || (batchCount >= INT_MAX) )
  {
    THError("Cublas_DgemmBatched only supports m, n, k, lda, ldb, ldc, batchCount"
            "with the bound [val] <= %d", INT_MAX);
  }

  adjustLdLevel3(transa, transb, m, n, k, &lda, &ldb, &ldc);
  hipblasOperation_t opa = convertTransToCublasOperation(transa);
  hipblasOperation_t opb = convertTransToCublasOperation(transb);

  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
  THCublasCheck(hipblasDgemmStridedBatched(handle,
                                   opa, opb, (int)m, (int)n, (int)k,
                                   &alpha, a, (int)lda, strideA, b, (int)ldb, strideB, &beta, c, (int)ldc, strideC,
                                   (int)batchCount));
}
#endif

