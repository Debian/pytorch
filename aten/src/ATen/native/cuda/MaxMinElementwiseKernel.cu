#include "hip/hip_runtime.h"
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/native/BinaryOps.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/zmath.cuh>

// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at { namespace native {

void max_elementwise_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == ScalarType::Bool) {
    gpu_kernel(iter, []GPU_LAMBDA(bool a, bool b) -> bool {
      return a || b;
    });
  } else if (isIntegralType(iter.dtype(), /*includeBool=*/ false)) {
    AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "max_elementwise_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        return ::max(a, b);
      });
    });
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "max_elementwise_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        // isnan(half) breaks the Windows build. We explicitly cast half to float.
        using acc_type = typename AccumulateType<scalar_t, /*is_cuda=*/true>::type;
        // We avoid using nan or nanf because we want to return the same type as scalar_t.
        if (::isnan(static_cast<acc_type>(a))) {
          return a;
        } else if (::isnan(static_cast<acc_type>(b))) {
          return b;
        } else {
          return ::max(a, b);
        }
      });
    });
  }
}

void min_elementwise_kernel_cuda(TensorIterator& iter) {
  if (iter.dtype() == ScalarType::Bool) {
    gpu_kernel(iter, []GPU_LAMBDA(bool a, bool b) -> bool {
      return a && b;
    });
  } else if (isIntegralType(iter.dtype(), /*includeBool=*/ false)) {
    AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "min_elementwise_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        return ::min(a, b);
      });
    });
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.dtype(), "min_elementwise_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
        // isnan(half) breaks the Windows build. We explicitly cast half to float.
        using acc_type = typename AccumulateType<scalar_t, /*is_cuda=*/true>::type;
        // We avoid using nan or nanf because we want to return the same type as scalar_t.
        if (::isnan(static_cast<acc_type>(a))) {
          return a;
        } else if (::isnan(static_cast<acc_type>(b))) {
          return b;
        } else {
          return ::min(a, b);
        }
      });
    });
  }
}

REGISTER_DISPATCH(max_elementwise_stub, &max_elementwise_kernel_cuda);
REGISTER_DISPATCH(min_elementwise_stub, &min_elementwise_kernel_cuda);

}} // namespace at::native
