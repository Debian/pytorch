#include "hip/hip_runtime.h"
#include <algorithm>

#include "caffe2/sgd/adagrad_fused_op_gpu.cuh"
#include "caffe2/utils/math.h"

namespace caffe2 {

namespace {

void inclusive_scan_wrapper(
    const int* length_data,
    int len_length,
    Tensor* temp_buffer,
    Tensor* prefix_sum_out,
    HIPContext* context_) {
  // Retrieve buffer size
  size_t temp_storage_bytes = 0;
  hipcub::DeviceScan::InclusiveSum(
      NULL,
      temp_storage_bytes,
      length_data,
      prefix_sum_out->template mutable_data<int>(),
      len_length,
      context_->hip_stream());
  // Allocate temporary storage
  auto buffer_size = (temp_storage_bytes + sizeof(int)) / sizeof(int);
  temp_buffer->Resize(buffer_size);
  void* d_temp_storage =
      static_cast<void*>(temp_buffer->template mutable_data<int>());
  // Run inclusive prefix sum
  hipcub::DeviceScan::InclusiveSum(
      d_temp_storage,
      temp_storage_bytes,
      length_data,
      prefix_sum_out->template mutable_data<int>(),
      len_length,
      context_->hip_stream());
}

template <typename SIndex, typename TParam, typename T, bool ExactBlock = false>
#ifdef __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_2(1024, SEGREDUCE_MINBLOCKS)
#endif
__global__ void sparse_adagrad_fused_length_sum_gradient_kernel(
    const int* __restrict__ prefix_sum_length_data, // prefix of lengths
                                                    // (offsets for the
                                                    // segments)
    int N, // number of rows (hash size) of embedding table
    int post, // embedding dimension size
    int len_length, // number of segments
    const float epsilon,
    TParam* param,
    TParam* param_mom,
    const SIndex* indices,
    const T* __restrict__ grad,
    const float* lr,
    float weight_decay = 0.f) {
  const float LR = lr[0];
  // len_length blocks, each block process one segment
  int group = blockIdx.x; // the group-th segment
  int start = group == 0
      ? 0
      : prefix_sum_length_data[group - 1]; // start offset of the segment
  int end = prefix_sum_length_data[group]; // end offset of the segment
  CUDA_KERNEL_ASSERT(start <= N);
  CUDA_KERNEL_ASSERT(end <= N);

  if (ExactBlock) {
    const size_t gradIdx = group * post + threadIdx.x; // index for grad
    for (int line = start + threadIdx.y; line < end; line += blockDim.y) {
      // line: the idx in the indices
      // threadIdx.x: index in the embedding dimension
      const SIndex index =
          indices[line]; // the index-th row in the embedding table
      const size_t paramIdx = index * post + threadIdx.x; // index for param

      float gi = grad[gradIdx] + weight_decay * param[paramIdx];

      float mom_new = gi * gi + param_mom[paramIdx];
      param_mom[paramIdx] = mom_new;
      float param_new = LR * gi / (sqrtf(mom_new) + epsilon) + param[paramIdx];
      param[paramIdx] = param_new;
    }
  } else {
    for (int i = threadIdx.x; i < post; i += blockDim.x) {
      // i: index in the embedding dimension
      const size_t gradIdx = group * post + i; // index for grad
      for (int line = start; line < end; ++line) {
        // line: the idx in the indices
        const SIndex index =
            indices[line]; // the index row in the embedding table
        const size_t paramIdx = index * post + i; // index for param

        float gi = grad[gradIdx] + weight_decay * param[paramIdx];

        float mom_new = gi * gi + param_mom[paramIdx];
        param_mom[paramIdx] = mom_new;
        float param_new =
            LR * gi / (sqrtf(mom_new) + epsilon) + param[paramIdx];
        param[paramIdx] = param_new;
      }
    }
  }
}

template <typename SIndex, typename TParam, typename T, int NumThreads>
#ifdef __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_2(1024, SEGREDUCE_MINBLOCKS)
#endif
__global__ void sparse_adagrad_fused_length_weighted_sum_gradient_kernel(
    const int* __restrict__ prefix_sum_length_data,
    int N, // number of rows (hash size) of embedding table
    int post, // embedding dimension size
    int len_length, // number of segments
    const float epsilon,
    TParam* param,
    TParam* param_mom,
    const SIndex* indices,
    const T* __restrict__ grad,
    const T* __restrict__ weights,
    T* __restrict__ weights_grad_out,
    const float* lr,
    float weight_decay = 0.f) {
  const float LR = lr[0];
  // len_length blocks, each block process one segment
  int group = blockIdx.x; // the group-th segment
  int start = group == 0
      ? 0
      : prefix_sum_length_data[group - 1]; // start offset of the segment
  int end = prefix_sum_length_data[group]; // end offset of the segment
  CUDA_KERNEL_ASSERT(start <= N);
  CUDA_KERNEL_ASSERT(end <= N);

  // TODO: Tuning NumThreads for w_grad
  typedef hipcub::BlockReduce<float, NumThreads> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  // TODO(jianyuhuang): parallelize this outer loop
  for (int line = start; line < end; ++line) {
    T w_grad = 0;
    // line: the idx in the indices
    const SIndex index =
        indices[line]; // the index-th row in the embedding table

    // SparseAdagradFusedWithSparseLengthsWeightedSumGradientOp also fuses
    // LengthsRangeFill + Gather operator. In the normal SLWS operator weight is
    // accessed via weights[line] but in most cases the weights are
    // generated by LengthsRangeFill and Gather operator.
    // For example, if lengths is [2, 3, 1] LengthsRangeFill will generate [0,
    // 1; 0, 1, 2; 0] and they are used as indices of Gather.
    // So if we fuse all of these, weights[line] just becomes
    // weights[line - start].
    auto in_weight_temp = weights[line - start];

    for (int i = threadIdx.x; i < post; i += blockDim.x) {
      // i: index in the embedding dimension
      const size_t gradIdx = group * post + i; // index for in_grad
      const size_t paramIdx = index * post + i; // index for param

      // TODO: trying to reduce the variable number (common subexpression
      // elimination).
      auto in_grad_temp = grad[gradIdx];
      w_grad += in_grad_temp * param[paramIdx];

      auto out_grad_temp =
          in_weight_temp * in_grad_temp + weight_decay * param[paramIdx];

      // TODO: split it into two kernels to make it more similar to exact fusion
      // kernel (not Approx on CPUs).
      float mom_new = out_grad_temp * out_grad_temp + param_mom[paramIdx];
      param_mom[paramIdx] = mom_new;
      float param_new =
          LR * out_grad_temp / (sqrtf(mom_new) + epsilon) + param[paramIdx];
      param[paramIdx] = param_new;
    }

    w_grad = BlockReduce(temp_storage).Reduce(w_grad, hipcub::Sum());

    if (threadIdx.x == 0) {
      weights_grad_out[line] = w_grad;
    }
    __syncthreads();
  }
}

template <typename SIndex, typename TParam, typename T, int NumThreads>
#ifdef __HIP_PLATFORM_HCC__
C10_LAUNCH_BOUNDS_2(1024, SEGREDUCE_MINBLOCKS)
#endif
__global__
    void rowwise_sparse_adagrad_fused_length_weighted_sum_gradient_kernel(
        const int* __restrict__ prefix_sum_length_data, // prefix of lengths
                                                        // (offsets for the
                                                        // segments)
        int N, // number of rows (hash size) of embedding table
        int post, // embedding dimension size
        int len_length, // number of segments
        const float epsilon,
        TParam* param,
        T* param_mom,
        const SIndex* indices,
        const T* __restrict__ grad,
        const T* __restrict__ weights,
        T* __restrict__ weights_grad_out,
        const float* lr,
        float weight_decay = 0.f) {
  const float LR = lr[0];
  // len_length blocks, each block process one segment
  int group = blockIdx.x; // the group-th segment
  int start = group == 0
      ? 0
      : prefix_sum_length_data[group - 1]; // start offset of the segment
  int end = prefix_sum_length_data[group]; // end offset of the segment
  CUDA_KERNEL_ASSERT(start <= N);
  CUDA_KERNEL_ASSERT(end <= N);

  // TODO: Tuning NumThreads for w_grad
  typedef hipcub::BlockReduce<float, NumThreads> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  int valid = min(post, blockDim.x);

  // for avg_square_weight. Can we reuse temp_storage
  __shared__ typename BlockReduce::TempStorage temp_storage2;
  // Why do we need to add typename here?

  // TODO(jianyuhuang): parallelize this outer loop
  for (int line = start; line < end; ++line) {
    T w_grad = 0;
    // i: index in the embedding dimension
    const SIndex index = indices[line];

    auto in_weight_temp = weights[line - start];

    float sum_squares = 0.0;
    __shared__ float row_sum_squares_avg;

    for (int i = threadIdx.x; i < post; i += blockDim.x) {
      const float x_ij =
          grad[group * post + i] + weight_decay * param[index * post + i];
      sum_squares += x_ij * x_ij;
    }
    float reduce_result = BlockReduce(temp_storage2).Sum(sum_squares, valid);

    if (threadIdx.x == 0) {
      row_sum_squares_avg = reduce_result / static_cast<float>(post);
      param_mom[index] +=
          static_cast<T>(row_sum_squares_avg * in_weight_temp * in_weight_temp);
    }
    __syncthreads();

    // update param
    float step = LR / (sqrtf(param_mom[index]) + epsilon);

    for (int i = threadIdx.x; i < post; i += blockDim.x) {
      const size_t gradIdx = group * post + i; // index for in_grad
      const size_t paramIdx = index * post + i; // index for param
      // TODO: trying to reduce the variable number (common subexpression
      // elimination).
      auto in_grad_temp = grad[gradIdx];
      w_grad += in_grad_temp * param[paramIdx];
      auto out_grad_temp =
          in_weight_temp * in_grad_temp + weight_decay * param[paramIdx];

      // TODO: split it into two kernels to make it more similar to exact
      // fusion kernel (not Approx on CPUs).
      param[paramIdx] = out_grad_temp * step + param[paramIdx];
    }
    w_grad = BlockReduce(temp_storage).Reduce(w_grad, hipcub::Sum());

    if (threadIdx.x == 0) {
      weights_grad_out[line] = w_grad;
    }
    __syncthreads();
  }
}

} // namespace

template <typename T, typename TLengths, class Context>
class CUDASparseAdagradFusedWithSparseLengthsSumGradientOp final
    : public Operator<Context> {
 public:
  USE_OPERATOR_CONTEXT_FUNCTIONS;
  CUDASparseAdagradFusedWithSparseLengthsSumGradientOp(
      const OperatorDef& operator_def,
      Workspace* ws)
      : Operator<Context>(operator_def, ws),
        epsilon_(this->template GetSingleArgument<float>("epsilon", 1e-5f)),
        weight_decay_(
            this->template GetSingleArgument<float>("weight_decay", 0.f)) {
    VLOG(1) << "gradient optimization operator in use: "
            << "CUDASparseAdagradFusedWithSparseLengthSumGradientOp"
            << " weight_decay_=" << weight_decay_;

    const T decay = this->template GetSingleArgument<T>("decay", 1.0f);
    CAFFE_ENFORCE_EQ(decay, 1.0, "Decay is not supported for SparseAdagradOp");
  }

  bool RunOnDevice() override {
    // Enforce shapes
    CAFFE_ENFORCE_EQ(Input(PARAM).size(), Input(MOMENT_1).size());
    CAFFE_ENFORCE_EQ(Input(LR).size(), 1);
    CAFFE_ENFORCE_EQ(
        Input(PARAM).size_from_dim(1),
        Input(GRAD).size_from_dim(Input(INDICES).ndim()));

    return DispatchHelper<TensorTypes<int32_t, int64_t>>::call(
        this, Input(INDICES));
  }

  template <typename IndexType>
  bool DoRunWithType() {
    auto n = Input(INDICES).size();
    if (n == 0) {
      return true;
    }
    return DispatchHelper<TensorTypes2<float, at::Half>, IndexType>::call(
        this, Input(PARAM));
  }

  template <typename IndexType, typename TParam>
  bool DoRunWithType2() {
    auto& segmentGradsInput = Input(GRAD);
    auto& lengthsInput = Input(LENGTHS);
    auto& indicesInput = Input(INDICES);

    CAFFE_ENFORCE_EQ(1, lengthsInput.dim(), "LENGTHS must be a vector");
    CAFFE_ENFORCE_GT(Input(GRAD).dim(), 0);

    // Enforce:
    // input(embedding/momentum) == outputs(embedding/momentum)
    CAFFE_ENFORCE_EQ(
        Input(PARAM).numel(),
        Input(MOMENT_1).numel(),
        "Input Param size: ",
        Input(PARAM).numel(),
        " Input Moment size: ",
        Input(MOMENT_1).numel());

    const int len_length = lengthsInput.dim(0);
    CAFFE_ENFORCE(segmentGradsInput.dim() > 0);
    CAFFE_ENFORCE(len_length == segmentGradsInput.dim(0));

    int output_0dim = indicesInput.dim(0);

    if (len_length <= 0) {
      // return early to avoid invalid empty kernel
      return true;
    }

    inclusive_scan_length_buffer_.ResizeLike(lengthsInput);
    inclusive_scan_wrapper(
        lengthsInput.template data<int>(),
        len_length,
        &inclusive_scan_buffer_,
        &inclusive_scan_length_buffer_,
        &context_);

    // compute output size using length
    auto* prefix_sum_length_data =
        inclusive_scan_length_buffer_.template data<int>();

    const auto* lr = Input(LR).template data<T>();
    const auto* indices = Input(INDICES).template data<IndexType>();
    const T* grad = Input(GRAD).template data<T>();
    auto* paramOut = Output(OUTPUT_PARAM)->template mutable_data<TParam>();
    auto* momentOut = Output(OUTPUT_MOMENT_1)->template mutable_data<TParam>();

    int N = output_0dim;
    int post = segmentGradsInput.size_from_dim(1);

    auto maxThreads =
        GetDeviceProperty(CaffeCudaGetDevice()).maxThreadsPerBlock;

    if (post <= maxThreads) {
      int multiple = std::min(maxThreads / post, SEGREDUCE_MINBLOCKS);
      dim3 block(post, multiple);

      // calling cuda kernel with ExactBlock = true
      // T should be rename as TGRAD ?
      sparse_adagrad_fused_length_sum_gradient_kernel<
          IndexType,
          TParam,
          T,
          true><<<len_length, block, 0, context_.hip_stream()>>>(
          prefix_sum_length_data,
          N,
          post,
          len_length,
          epsilon_,
          paramOut,
          momentOut,
          indices,
          grad,
          lr,
          weight_decay_);
    } else {
      // calling cuda kernel with ExactBlock = false
      sparse_adagrad_fused_length_sum_gradient_kernel<
          IndexType,
          TParam,
          T,
          false><<<len_length, maxThreads, 0, context_.hip_stream()>>>(
          prefix_sum_length_data,
          N,
          post,
          len_length,
          epsilon_,
          paramOut,
          momentOut,
          indices,
          grad,
          lr,
          weight_decay_);
    }
    return true;
  }

 private:
  // menber field to manage memory
  Tensor inclusive_scan_buffer_{CUDA};
  Tensor inclusive_scan_length_buffer_{CUDA};

 protected:
  T epsilon_;
  T weight_decay_;
  INPUT_TAGS(PARAM, MOMENT_1, INDICES, GRAD, LR, LENGTHS);
  OUTPUT_TAGS(OUTPUT_PARAM, OUTPUT_MOMENT_1);
};

template <typename T, typename TLengths, class Context>
class CUDASparseAdagradFusedWithSparseLengthsWeightedSumGradientOp final
    : public Operator<Context> {
 public:
  USE_OPERATOR_CONTEXT_FUNCTIONS;
  CUDASparseAdagradFusedWithSparseLengthsWeightedSumGradientOp(
      const OperatorDef& operator_def,
      Workspace* ws)
      : Operator<Context>(operator_def, ws),
        epsilon_(this->template GetSingleArgument<float>("epsilon", 1e-5f)),
        weight_decay_(
            this->template GetSingleArgument<float>("weight_decay", 0.f)) {
    VLOG(1) << "gradient optimization operator in use: "
            << "CUDASparseAdagradFusedWithSparseLengthWeightedSumGradientOp"
            << " weight_decay_=" << weight_decay_;

    const T decay = this->template GetSingleArgument<T>("decay", 1.0f);
    CAFFE_ENFORCE_EQ(decay, 1.0, "Decay is not supported for SparseAdagradOp");
  }

  bool RunOnDevice() override {
    // Enforce shapes
    CAFFE_ENFORCE_EQ(Input(PARAM).size(), Input(MOMENT_1).size());
    CAFFE_ENFORCE_EQ(Input(LR).size(), 1);
    CAFFE_ENFORCE_EQ(
        Input(PARAM).size_from_dim(1),
        Input(GRAD).size_from_dim(Input(INDICES).ndim()));

    return DispatchHelper<TensorTypes<int32_t, int64_t>>::call(
        this, Input(INDICES));
  }

  template <typename IndexType>
  bool DoRunWithType() {
    auto n = Input(INDICES).size();
    if (n == 0) {
      // Allocate output to an empty tensor
      Output(AUX_GRAD, n, at::dtype<T>());
      return true;
    }
    return DispatchHelper<TensorTypes2<float, at::Half>, IndexType>::call(
        this, Input(PARAM));
  }

  template <typename IndexType, typename TParam>
  bool DoRunWithType2() {
    auto& segmentGradsInput = Input(GRAD);
    auto& lengthsInput = Input(LENGTHS);
    auto& indicesInput = Input(INDICES);
    auto& weightsInput = Input(AUX_PARAM);

    CAFFE_ENFORCE_EQ(1, lengthsInput.dim(), "LENGTHS must be a vector");
    CAFFE_ENFORCE_EQ(1, weightsInput.dim(), "WEIGHTS must be a vector");
    CAFFE_ENFORCE_GT(Input(GRAD).dim(), 0);

    // Enforce:
    // input(embedding/momentum) == outputs(embedding/momentum)
    CAFFE_ENFORCE_EQ(
        Input(PARAM).numel(),
        Input(MOMENT_1).numel(),
        "Input Param size: ",
        Input(PARAM).numel(),
        " Input Moment size: ",
        Input(MOMENT_1).numel());

    const int len_length = lengthsInput.dim(0);
    CAFFE_ENFORCE(segmentGradsInput.dim() > 0);
    CAFFE_ENFORCE(len_length == segmentGradsInput.dim(0));

    int output_0dim = indicesInput.dim(0);
    auto* weightGradsOutput =
        Output(AUX_GRAD, indicesInput.sizes(), at::dtype<T>());

    T* out_weight_grads = weightGradsOutput->template mutable_data<T>();

    if (len_length <= 0) {
      // return early to avoid invalid empty kernel
      return true;
    }

    inclusive_scan_length_buffer_.ResizeLike(lengthsInput);
    inclusive_scan_wrapper(
        lengthsInput.template data<int>(),
        len_length,
        &inclusive_scan_buffer_,
        &inclusive_scan_length_buffer_,
        &context_);

    // compute output size using length
    auto* prefix_sum_length_data =
        inclusive_scan_length_buffer_.template data<int>();

    const auto* lr = Input(LR).template data<T>();
    const auto* indices = Input(INDICES).template data<IndexType>();
    const T* grad = Input(GRAD).template data<T>();
    const T* weights = weightsInput.template data<T>();
    auto* paramOut = Output(OUTPUT_PARAM)->template mutable_data<TParam>();
    auto* momentOut = Output(OUTPUT_MOMENT_1)->template mutable_data<TParam>();

    int N = output_0dim;
    int post = segmentGradsInput.size_from_dim(1);

    auto maxThreads =
        GetDeviceProperty(CaffeCudaGetDevice()).maxThreadsPerBlock;

    if (post > 128) {
      sparse_adagrad_fused_length_weighted_sum_gradient_kernel<
          IndexType,
          TParam,
          T,
          512><<<len_length, 512, 0, context_.hip_stream()>>>(
          prefix_sum_length_data,
          N,
          post,
          len_length,
          epsilon_,
          paramOut,
          momentOut,
          indices,
          grad,
          weights,
          out_weight_grads,
          lr,
          weight_decay_);
    } else if (post > 64) {
      sparse_adagrad_fused_length_weighted_sum_gradient_kernel<
          IndexType,
          TParam,
          T,
          128><<<len_length, 128, 0, context_.hip_stream()>>>(
          prefix_sum_length_data,
          N,
          post,
          len_length,
          epsilon_,
          paramOut,
          momentOut,
          indices,
          grad,
          weights,
          out_weight_grads,
          lr,
          weight_decay_);
    } else if (post > 32) {
      sparse_adagrad_fused_length_weighted_sum_gradient_kernel<
          IndexType,
          TParam,
          T,
          64><<<len_length, 64, 0, context_.hip_stream()>>>(
          prefix_sum_length_data,
          N,
          post,
          len_length,
          epsilon_,
          paramOut,
          momentOut,
          indices,
          grad,
          weights,
          out_weight_grads,
          lr,
          weight_decay_);
    } else {
      sparse_adagrad_fused_length_weighted_sum_gradient_kernel<
          IndexType,
          TParam,
          T,
          32><<<len_length, 32, 0, context_.hip_stream()>>>(
          prefix_sum_length_data,
          N,
          post,
          len_length,
          epsilon_,
          paramOut,
          momentOut,
          indices,
          grad,
          weights,
          out_weight_grads,
          lr,
          weight_decay_);
    }
    return true;
  }

 private:
  // menber field to manage memory
  Tensor inclusive_scan_buffer_{CUDA};
  Tensor inclusive_scan_length_buffer_{CUDA};

 protected:
  T epsilon_;
  T weight_decay_;
  INPUT_TAGS(PARAM, MOMENT_1, AUX_PARAM, INDICES, GRAD, LR, LENGTHS);
  OUTPUT_TAGS(OUTPUT_PARAM, OUTPUT_MOMENT_1, AUX_GRAD);
};

template <typename T, typename TLengths, class Context>
class CUDARowWiseSparseAdagradFusedWithSparseLengthsSumGradientOp final
    : public Operator<Context> {
 public:
  USE_OPERATOR_CONTEXT_FUNCTIONS;
  CUDARowWiseSparseAdagradFusedWithSparseLengthsSumGradientOp(
      const OperatorDef& operator_def,
      Workspace* ws)
      : Operator<Context>(operator_def, ws),
        epsilon_(this->template GetSingleArgument<float>("epsilon", 1e-5f)),
        weight_decay_(
            this->template GetSingleArgument<float>("weight_decay", 0.f)) {
    VLOG(1) << "gradient optimization operator in use: "
            << "CUDARowWiseSparseAdagradFusedWithSparseLengthSumGradientOp"
            << " weight_decay_=" << weight_decay_;

    const T decay = this->template GetSingleArgument<T>("decay", 1.0f);
    CAFFE_ENFORCE_EQ(decay, 1.0, "Decay is not supported for SparseAdagradOp");
  }

  bool RunOnDevice() override {
    // Enforce shapes
    CAFFE_ENFORCE_EQ(Input(LR).size(), 1);
    CAFFE_ENFORCE_EQ(
        Input(PARAM).size_from_dim(1),
        Input(GRAD).size_from_dim(Input(INDICES).ndim()));

    return DispatchHelper<TensorTypes<int32_t, int64_t>>::call(
        this, Input(INDICES));
  }

  template <typename IndexType>
  bool DoRunWithType() {
    auto n = Input(INDICES).size();
    if (n == 0) {
      return true;
    }
    return DispatchHelper<TensorTypes2<float, at::Half>, IndexType>::call(
        this, Input(PARAM));
  }

  template <typename IndexType, typename TParam>
  bool DoRunWithType2() {
    auto& segmentGradsInput = Input(GRAD);
    auto& lengthsInput = Input(LENGTHS);
    auto& indicesInput = Input(INDICES);

    CAFFE_ENFORCE_EQ(1, lengthsInput.dim(), "LENGTHS must be a vector");
    CAFFE_ENFORCE_GT(Input(GRAD).dim(), 0);

    // Enforce:
    // number of rows: input(embedding/momentum) ==
    // outputs(embedding/momentum)
    CAFFE_ENFORCE_EQ(
        Input(PARAM).dim(0),
        Input(MOMENT_1).dim(0),
        "Input Param number of rows: ",
        Input(PARAM).dim(0),
        " Input Moment size: ",
        Input(MOMENT_1).dim(0));

    const int len_length = lengthsInput.dim(0);
    CAFFE_ENFORCE(segmentGradsInput.dim() > 0);
    CAFFE_ENFORCE(len_length == segmentGradsInput.dim(0));

    int output_0dim = indicesInput.dim(0);

    if (len_length <= 0) {
      // return early to avoid invalid empty kernel
      return true;
    }

    inclusive_scan_length_buffer_.ResizeLike(lengthsInput);
    inclusive_scan_wrapper(
        lengthsInput.template data<int>(),
        len_length,
        &inclusive_scan_buffer_,
        &inclusive_scan_length_buffer_,
        &context_);

    // compute output size using length
    auto* prefix_sum_length_data =
        inclusive_scan_length_buffer_.template data<int>();

    const auto* lr = Input(LR).template data<T>();
    const auto* indices = Input(INDICES).template data<IndexType>();
    const T* grad = Input(GRAD).template data<T>();
    auto* paramOut = Output(OUTPUT_PARAM)->template mutable_data<TParam>();
    auto* momentOut = Output(OUTPUT_MOMENT_1)->template mutable_data<T>();

    int N = output_0dim;
    int post = segmentGradsInput.size_from_dim(1);

    auto maxThreads =
        GetDeviceProperty(CaffeCudaGetDevice()).maxThreadsPerBlock;

    if (post <= maxThreads / 2 && post % 32 == 0) {
      // Fast path when the embedding dimension is a multiple of 32, using
      // WarpReduce.
      int multiple = std::min(maxThreads / post, SEGREDUCE_MINBLOCKS);
      dim3 block(post, multiple);

      rowwise_sparse_adagrad_fused_length_sum_gradient_kernel<
          IndexType,
          TParam,
          T,
          true><<<len_length, block, 0, context_.hip_stream()>>>(
          prefix_sum_length_data,
          N,
          post,
          len_length,
          epsilon_,
          paramOut,
          momentOut,
          indices,
          grad,
          lr,
          weight_decay_);
    } else {
      rowwise_sparse_adagrad_fused_length_sum_gradient_kernel<
          IndexType,
          TParam,
          T,
          false>
          <<<len_length,
             std::min(maxThreads, post),
             0,
             context_.hip_stream()>>>(
              prefix_sum_length_data,
              N,
              post,
              len_length,
              epsilon_,
              paramOut,
              momentOut,
              indices,
              grad,
              lr,
              weight_decay_);
    }

    return true;
  }

 private:
  // menber field to manage memory
  Tensor inclusive_scan_buffer_{CUDA};
  Tensor inclusive_scan_length_buffer_{CUDA};

 protected:
  T epsilon_;
  T weight_decay_;
  INPUT_TAGS(PARAM, MOMENT_1, INDICES, GRAD, LR, LENGTHS);
  OUTPUT_TAGS(OUTPUT_PARAM, OUTPUT_MOMENT_1);
};

template <typename T, typename TLengths, class Context>
class CUDARowWiseSparseAdagradFusedWithSparseLengthsWeightedSumGradientOp final
    : public Operator<Context> {
 public:
  USE_OPERATOR_CONTEXT_FUNCTIONS;
  CUDARowWiseSparseAdagradFusedWithSparseLengthsWeightedSumGradientOp(
      const OperatorDef& operator_def,
      Workspace* ws)
      : Operator<Context>(operator_def, ws),
        epsilon_(this->template GetSingleArgument<float>("epsilon", 1e-5f)),
        weight_decay_(
            this->template GetSingleArgument<float>("weight_decay", 0.f)) {
    VLOG(1)
        << "gradient optimization operator in use: "
        << "CUDARowWiseSparseAdagradFusedWithSparseLengthWeightedSumGradientOp"
        << " weight_decay_=" << weight_decay_;

    const T decay = this->template GetSingleArgument<T>("decay", 1.0f);
    CAFFE_ENFORCE_EQ(decay, 1.0, "Decay is not supported for SparseAdagradOp");
  }

  bool RunOnDevice() override {
    // Enforce shapes
    CAFFE_ENFORCE_EQ(Input(LR).size(), 1);
    CAFFE_ENFORCE_EQ(
        Input(PARAM).size_from_dim(1),
        Input(GRAD).size_from_dim(Input(INDICES).ndim()));

    return DispatchHelper<TensorTypes<int32_t, int64_t>>::call(
        this, Input(INDICES));
  }

  template <typename IndexType>
  bool DoRunWithType() {
    auto n = Input(INDICES).size();
    if (n == 0) {
      Output(AUX_GRAD, n, at::dtype<T>());
      return true;
    }
    return DispatchHelper<TensorTypes2<float, at::Half>, IndexType>::call(
        this, Input(PARAM));
  }

  template <typename IndexType, typename TParam>
  bool DoRunWithType2() {
    auto& segmentGradsInput = Input(GRAD);
    auto& lengthsInput = Input(LENGTHS);
    auto& indicesInput = Input(INDICES);
    auto& weightsInput = Input(AUX_PARAM);

    CAFFE_ENFORCE_EQ(1, lengthsInput.dim(), "LENGTHS must be a vector");
    CAFFE_ENFORCE_EQ(1, weightsInput.dim(), "WEIGHTS must be a vector");
    CAFFE_ENFORCE_GT(Input(GRAD).dim(), 0);

    // Enforce:
    // number of rows: input(embedding/momentum) ==
    // outputs(embedding/momentum)
    CAFFE_ENFORCE_EQ(
        Input(PARAM).dim(0),
        Input(MOMENT_1).dim(0),
        "Input Param number of rows: ",
        Input(PARAM).dim(0),
        " Input Moment size: ",
        Input(MOMENT_1).dim(0));

    const int len_length = lengthsInput.dim(0);
    CAFFE_ENFORCE(segmentGradsInput.dim() > 0);
    CAFFE_ENFORCE(len_length == segmentGradsInput.dim(0));

    int output_0dim = indicesInput.dim(0);
    auto* weightGradsOutput =
        Output(AUX_GRAD, indicesInput.sizes(), at::dtype<T>());

    T* out_weight_grads = weightGradsOutput->template mutable_data<T>();

    if (len_length <= 0) {
      // return early to avoid invalid empty kernel
      return true;
    }

    inclusive_scan_length_buffer_.ResizeLike(lengthsInput);
    inclusive_scan_wrapper(
        lengthsInput.template data<int>(),
        len_length,
        &inclusive_scan_buffer_,
        &inclusive_scan_length_buffer_,
        &context_);

    // compute output size using length
    auto* prefix_sum_length_data =
        inclusive_scan_length_buffer_.template data<int>();

    const auto* lr = Input(LR).template data<T>();
    const auto* indices = Input(INDICES).template data<IndexType>();
    const T* grad = Input(GRAD).template data<T>();
    const T* weights = weightsInput.template data<T>();
    auto* paramOut = Output(OUTPUT_PARAM)->template mutable_data<TParam>();
    auto* momentOut = Output(OUTPUT_MOMENT_1)->template mutable_data<T>();

    int N = output_0dim;
    int post = segmentGradsInput.size_from_dim(1);

    auto maxThreads =
        GetDeviceProperty(CaffeCudaGetDevice()).maxThreadsPerBlock;

    if (post > 128) {
      rowwise_sparse_adagrad_fused_length_weighted_sum_gradient_kernel<
          IndexType,
          TParam,
          T,
          512><<<len_length, 512, 0, context_.hip_stream()>>>(
          prefix_sum_length_data,
          N,
          post,
          len_length,
          epsilon_,
          paramOut,
          momentOut,
          indices,
          grad,
          weights,
          out_weight_grads,
          lr,
          weight_decay_);
    } else if (post > 64) {
      rowwise_sparse_adagrad_fused_length_weighted_sum_gradient_kernel<
          IndexType,
          TParam,
          T,
          128><<<len_length, 128, 0, context_.hip_stream()>>>(
          prefix_sum_length_data,
          N,
          post,
          len_length,
          epsilon_,
          paramOut,
          momentOut,
          indices,
          grad,
          weights,
          out_weight_grads,
          lr,
          weight_decay_);
    } else if (post > 32) {
      rowwise_sparse_adagrad_fused_length_weighted_sum_gradient_kernel<
          IndexType,
          TParam,
          T,
          64><<<len_length, 64, 0, context_.hip_stream()>>>(
          prefix_sum_length_data,
          N,
          post,
          len_length,
          epsilon_,
          paramOut,
          momentOut,
          indices,
          grad,
          weights,
          out_weight_grads,
          lr,
          weight_decay_);
    } else {
      rowwise_sparse_adagrad_fused_length_weighted_sum_gradient_kernel<
          IndexType,
          TParam,
          T,
          32><<<len_length, 32, 0, context_.hip_stream()>>>(
          prefix_sum_length_data,
          N,
          post,
          len_length,
          epsilon_,
          paramOut,
          momentOut,
          indices,
          grad,
          weights,
          out_weight_grads,
          lr,
          weight_decay_);
    }

    return true;
  }

 private:
  // menber field to manage memory
  Tensor inclusive_scan_buffer_{CUDA};
  Tensor inclusive_scan_length_buffer_{CUDA};

 protected:
  T epsilon_;
  T weight_decay_;
  INPUT_TAGS(PARAM, MOMENT_1, AUX_PARAM, INDICES, GRAD, LR, LENGTHS);
  OUTPUT_TAGS(OUTPUT_PARAM, OUTPUT_MOMENT_1, AUX_GRAD);
};

// For GPU, the implementation of the exact and approx (RowWise)SparseAdagrad
// fusion are both approximate implementations.
// When we don't have the duplicated indices, the outputs are the same as the
// CPU implementation.
REGISTER_HIP_OPERATOR(
    SparseAdagradFusedWithSparseLengthsSumGradient,
    CUDASparseAdagradFusedWithSparseLengthsSumGradientOp<
        float,
        int,
        HIPContext>);
REGISTER_HIP_OPERATOR(
    SparseAdagradFusedWithSparseLengthsWeightedSumGradient,
    CUDASparseAdagradFusedWithSparseLengthsWeightedSumGradientOp<
        float,
        int,
        HIPContext>);
REGISTER_HIP_OPERATOR(
    SparseAdagradFusedWithSparseLengthsWeightedSumGradientApprox,
    CUDASparseAdagradFusedWithSparseLengthsWeightedSumGradientOp<
        float,
        int,
        HIPContext>);

REGISTER_HIP_OPERATOR(
    RowWiseSparseAdagradFusedWithSparseLengthsSumGradient,
    CUDARowWiseSparseAdagradFusedWithSparseLengthsSumGradientOp<
        float,
        int,
        HIPContext>);
REGISTER_HIP_OPERATOR(
    RowWiseSparseAdagradFusedWithSparseLengthsWeightedSumGradient,
    CUDARowWiseSparseAdagradFusedWithSparseLengthsWeightedSumGradientOp<
        float,
        int,
        HIPContext>);
REGISTER_HIP_OPERATOR(
    RowWiseSparseAdagradFusedWithSparseLengthsWeightedSumGradientApprox,
    CUDARowWiseSparseAdagradFusedWithSparseLengthsWeightedSumGradientOp<
        float,
        int,
        HIPContext>);

} // namespace caffe2

#undef SEGREDUCE_MINBLOCKS
